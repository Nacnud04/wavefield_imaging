
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

// macro for 1d index to simulate a 3d matrix
#define INDEX3D(ix, iy, iz, nx, nz) ((ix)+(iz)*(nx)+(iy)*(nz)*(nx))

void expand_cpu_3d(float *a, float *b, int nb, int x_a, int x_b, int y_a, int y_b, int z_a, int z_b){

        // copy into other array
	for (int ix = 0; ix < x_a; ix++) {
                for (int iy = 0; iy < y_a; iy++) {
                        for (int iz = 0; iz < z_a; iz++) {
                                b[INDEX3D(ix+nb,iy+nb,iz+nb,z_b,x_b)] = a[INDEX3D(ix,iy,iz,x_a,z_a)];
                        }
                }
        }
        // expand z direction
        
	for (int ix = 0; ix < x_b; ix++) {
                for (int iy = 0; iy < y_b; iy++) {
                        for (int iz = 0; iz < nb; iz++) {
                                b[INDEX3D(ix,iy,iz,x_b,z_b)] = b[INDEX3D(ix,iy,nb,x_b,z_b)];
                                b[INDEX3D(ix,iy,z_b-iz-1,x_b,z_b)] = b[INDEX3D(ix,iy,z_b-nb-1,x_b,z_b)];
                        }
                }
        }
	
        // expand y direction
        
	for (int ix = 0; ix < x_b; ix++) {
                for (int iy = 0; iy < nb; iy++) {
                        for (int iz = 0; iz < z_b; iz++) {
                                b[INDEX3D(ix,iy,iz,x_b,z_b)] = b[INDEX3D(ix,nb,iz,x_b,z_b)];
                                b[INDEX3D(ix,y_b-iy-1,iz,x_b,z_b)] = b[INDEX3D(ix,y_b-nb-1,iz,x_b,z_b)];
                        }
                }
        }
	
        //expand x direction
        
	for (int ix = 0; ix < nb; ix++) {
                for (int iy = 0; iy < y_b; iy++) {
                        for (int iz = 0; iz < z_b; iz++) {
                                b[INDEX3D(ix,iy,iz,x_b,z_b)] = b[INDEX3D(nb,iy,iz,x_b,z_b)];
                                b[INDEX3D(x_b-ix-1,iy,iz,x_b,z_b)] = b[INDEX3D(x_b-nb-1,iy,iz,x_b,z_b)];
                        }
                }
        }
	
}

__global__ void lint3d_bell_gpu(float *d_uu, float *d_ww, float *d_Sw000, float *d_Sw001, float *d_Sw010, float *d_Sw011, float *d_Sw100, float *d_Sw101, float *d_Sw110, float *d_Sw111, float *d_bell, int *d_jz, int *d_jy, int *d_jx, int it, int nc, int ns, int c, int nbell, int nxpad, int nzpad) {

        int ix = threadIdx.x;
        int iy = threadIdx.y;
        int iz = threadIdx.z;
        int ia = blockIdx.x;

        float wa = d_ww[it * nc * ns + c * ns + ia] * d_bell[(iy * (2*nbell+1)*(2*nbell+1)) + (iz * (2*nbell+1)) + ix];

        int y_comp = (d_jy[ia] - nbell) + iy;
	int z_comp = (d_jz[ia] - nbell) + iz;
	int x_comp = (d_jx[ia] - nbell) + ix;
	int xz = nxpad * nzpad;

        atomicAdd(&d_uu[(y_comp)     * xz + (z_comp)     * nxpad + (x_comp)], ((wa * d_Sw000[ia])));
        atomicAdd(&d_uu[(y_comp)     * xz + (z_comp + 1) * nxpad + (x_comp)], ((wa * d_Sw001[ia])));
        atomicAdd(&d_uu[(y_comp)     * xz + (z_comp)     * nxpad + (x_comp)], ((wa * d_Sw010[ia])));
        atomicAdd(&d_uu[(y_comp)     * xz + (z_comp + 1) * nxpad + (x_comp)], ((wa * d_Sw011[ia])));
        atomicAdd(&d_uu[(y_comp + 1) * xz + (z_comp)     * nxpad + (x_comp)], ((wa * d_Sw100[ia])));
        atomicAdd(&d_uu[(y_comp + 1) * xz + (z_comp + 1) * nxpad + (x_comp)], ((wa * d_Sw101[ia])));
        atomicAdd(&d_uu[(y_comp + 1) * xz + (z_comp)     * nxpad + (x_comp)], ((wa * d_Sw110[ia])));
        atomicAdd(&d_uu[(y_comp + 1) * xz + (z_comp + 1) * nxpad + (x_comp)], ((wa * d_Sw111[ia])));

}


// divergence 3d for cpml
#define NOP 4 // half of the order in space

__global__ void solve(float *d_fpo, float *d_po, float *d_ppo, float *d_vel,
		      float dx, float dy, float dz, float dt,
		      int nxpad, int nypad, int nzpad) {

	int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int iy = threadIdx.y + blockIdx.y * blockDim.y;
	int iz = threadIdx.z + blockIdx.z * blockDim.z;

	if (ix < nxpad && iy < nypad && iz < nzpad){
		
		int globalAddr = iy * nzpad * nxpad + iz * nxpad + ix;
		float pxx, pyy, pzz;
		float laplace;

		// perform only in boundaries:
		if (ix >= NOP && ix < nxpad-NOP && iy >= NOP && iy < nypad-NOP && iz >= NOP && iz < nzpad - NOP) {
			pxx = 0.; pyy = 0.; pzz = 0.;

			// calculate laplacian via finite differences
			pxx =    d_po[INDEX3D(ix+1, iy  , iz  ,nxpad,nzpad)] \
			      -2*d_po[INDEX3D(ix  , iy  , iz  ,nxpad,nzpad)] \
			        +d_po[INDEX3D(ix-1, iy  , iz  ,nxpad,nzpad)];
			pxx = pxx / (dx * dx);

			pyy =    d_po[INDEX3D(ix  , iy+1, iz  ,nxpad,nzpad)] \
                              -2*d_po[INDEX3D(ix  , iy  , iz  ,nxpad,nzpad)] \
                                +d_po[INDEX3D(ix  , iy-1, iz  ,nxpad,nzpad)];
			pyy = pyy / (dy * dy);

			pzz =    d_po[INDEX3D(ix  , iy  , iz+1,nxpad,nzpad)] \
                              -2*d_po[INDEX3D(ix  , iy  , iz  ,nxpad,nzpad)] \
                                +d_po[INDEX3D(ix  , iy  , iz-1,nxpad,nzpad)];
			pzz = pzz / (dz * dz);

			laplace = pxx + pyy + pzz;
	
		} else {
			laplace = 1.;
		}

		// compute pressure at next time step
		d_fpo[globalAddr] = (dt*dt)*(d_vel[globalAddr]*laplace) + 2*d_po[globalAddr] - d_ppo[globalAddr];

	}

}


__global__ void shift(float *d_fpo, float *d_po, float *d_ppo,
		      int nxpad, int nypad, int nzpad) {
	
	int ix = threadIdx.x + blockIdx.x * blockDim.x;
        int iy = threadIdx.y + blockIdx.y * blockDim.y;
        int iz = threadIdx.z + blockIdx.z * blockDim.z;

	if (ix < nxpad && iy < nypad && iz < nzpad){

		int globalAddr = iy * nzpad * nxpad + iz * nxpad + ix;
		
		// replace ppo with po and fpo with po
		d_ppo[globalAddr] = d_po[globalAddr];
		d_po[globalAddr] = d_fpo[globalAddr];

	}
}



__global__ void lint3d_extract_gpu(float *d_dd_pp, 
				   int it, int nr,
				   int nxpad, int nypad, int nzpad,
				   float *d_po, int *d_Rjx, int *d_Rjy, int *d_Rjz,
				   float *d_Rw000, float *d_Rw001, float *d_Rw010, float *d_Rw011, 
				   float *d_Rw100, float *d_Rw101, float *d_Rw110, float *d_Rw111) {

	int rr = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = it * nr;

	if (rr < nr){
		int y_comp = d_Rjy[rr] * nxpad * nzpad;
		int y_comp_1 = (d_Rjy[rr]+1) * nxpad * nzpad;
		int z_comp = d_Rjz[rr] * nxpad;
		int z_comp_1 = (d_Rjz[rr]+1) * nxpad;
		d_dd_pp[offset + rr] = d_po[y_comp   + z_comp   + (d_Rjx[rr])]   * d_Rw000[rr] +
                                       d_po[y_comp   + z_comp_1 + d_Rjx[rr]]     * d_Rw001[rr] +
                                       d_po[y_comp   + z_comp   + (d_Rjx[rr]+1)] * d_Rw010[rr] +
                                       d_po[y_comp   + z_comp_1 + (d_Rjx[rr]+1)] * d_Rw011[rr] +
                                       d_po[y_comp_1 + z_comp   + (d_Rjx[rr])]   * d_Rw100[rr] +
                                       d_po[y_comp_1 + z_comp_1 + d_Rjx[rr]]     * d_Rw101[rr] +
                                       d_po[y_comp_1 + z_comp   + (d_Rjx[rr]+1)] * d_Rw110[rr] +
                                       d_po[y_comp_1 + z_comp_1 + (d_Rjx[rr]+1)] * d_Rw111[rr];

	}

}


__global__ void freeSurf(float *d_po, int nxpad, int nypad, int nzpad, int nb) {

        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;
        int z = threadIdx.z + blockIdx.z * blockDim.z;

	if (x < nxpad && y < nypad && z < nb) {
		
		int addr = y * nxpad * nzpad + z * nxpad + x;

		d_po[addr] = 0;

	}
}


__global__ void spongeKernel(float *d_po, int nxpad, int nypad, int nzpad, int nb){

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int z = threadIdx.z + blockIdx.z * blockDim.z;

	float alpha = 0.90;

	// apply sponge
	if (x < nxpad && y < nypad && z < nzpad) {
        	
		// apply to low values
		if (x < nb || y < nb){

			int addr = y * nxpad * nzpad + z * nxpad + x;

			int i = nb - x;
			// dampining funct 1
			double damp = exp(-1.0*fabs(((i-1.0)*log(alpha))/nb)); 
			
			// dampining funct 2
			//double damp = exp(-1.0*fabs((pow((i-1.0),2)*log(alpha))/(pow(nb,2))));

			d_po[addr] *= damp;
		
		}
		// apply to high values
		if (x > nxpad - nb || y > nypad - nb || z > nzpad - nb) {
			
			int addr = y * nxpad * nzpad + z * nxpad + x;
			
			int i = x - (nxpad - nb);
			// dampining funct 1
			double damp = exp(-1.0*fabs(((i-1.0)*log(alpha))/nb));

                        // dampining funct 2
                        //double damp = exp(-1.0*fabs((pow((i-1.0),2)*log(alpha))/(pow(nb,2))));

			d_po[addr] *= damp;

		}

	}

}
