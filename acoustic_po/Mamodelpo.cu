#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

extern "C" {
    #include <rsf.h>
}

#include "fdutil.c"
#include "emodel3d_kernels.cu"

#define MIN(x, y) (((x) < (y)) ? (x): (y))
#define NOP 4

// funct to check gpu error
static void sf_check_gpu_error (const char *msg) {
    hipError_t err = hipGetLastError ();
    if (hipSuccess != err)
	sf_error ("Cuda error: %s: %s", msg, hipGetErrorString (err));
}

// entry
int main(int argc, char*argv[]) {

    // define input variables from sconstruct
    bool verb, frsf, snap, ssou, dabc;
    int jsnap, jdata;

    // define IO files
    sf_file Fwav=NULL; //wavelet
    sf_file Fsou=NULL; //source
    sf_file Frec=NULL; //receivers
    sf_file Fvel=NULL; //velocity
    sf_file Fden=NULL; //density
    sf_file Fdat=NULL; //data

}

