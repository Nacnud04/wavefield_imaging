#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

extern "C" {
    #include <rsf.h>
}

#include "fdutil_old.c"
#include "cart_kernels.cu"

#define MIN(x, y) (((x) < (y)) ? (x): (y))
#define NOP 4

// funct to check gpu error:
static void sf_check_gpu_error (const char *msg) {
    hipError_t err = hipGetLastError ();
    if (hipSuccess != err)
        sf_error ("Cuda error: %s: %s", msg, hipGetErrorString (err));
}

// entry
int main(int argc, char*argv[]) {
    
    // define input vars from scons
    bool verb, fsrf, snap, bnds, ssou, dabc;
    int jsnap, jdata;
    
    // define IO files
    sf_file Fwav=NULL; // wavelet
    sf_file Fsou=NULL; // sources
    sf_file Frec=NULL; // receviers
    sf_file Fvel=NULL; // velocity
    sf_file Fdat=NULL; // data
    sf_file Fwfl=NULL; // wavefield    	

    // define axis
    sf_axis at, awt, ax, ay, az, acx, acy, acz;
    sf_axis as, ar; // source, receiver, dimensions
		
    // define dimension sizes
    int nt, nz, ny, nx, ns, nr, ncs, nb;
    int it, ix, iy, iz;
    float dt, dz, dy, dx;

    // FDM structure
    fdm3d fdm=NULL;

    // device and host velocity
    float *h_vel, *d_vel;

    float *h_po, *d_fpo, *d_po, *d_ppo; // pressure
    float ***po=NULL;
    float ***oslice=NULL;

    // linear interpolation weights/indicies
    lint3d cs, cr;

    int nbell; // gaussian bell

    sf_init(argc, argv);

    // exec flags
    if(! sf_getbool("verb",&verb)) verb=false; /* verbosity flag */
    if(! sf_getbool("snap",&snap)) snap=false; /* wavefield snapshots flag */
    if(! sf_getbool("bnds",&bnds)) bnds=false; /* extract boundries of wavefield flag*/
    if(! sf_getbool("free",&fsrf)) fsrf=false; /* free surface flag */
    if(! sf_getbool("ssou",&ssou)) ssou=false; /* stress source */
    if(! sf_getbool("dabc",&dabc)) dabc=false; /* absorbing BC */
    sf_warning("verb:%b | snap:%b | free:%b | ssou:%b | dabc:%b",verb,snap,fsrf,ssou,dabc);

    // IO
    Fwav = sf_input ("in" ); /* wavelet   */
    Fvel = sf_input ("vel"); /* stiffness */
    Fsou = sf_input ("sou"); /* sources   */
    Frec = sf_input ("rec"); /* receivers */

    Fdat = sf_output("out"); // data
    Fwfl = sf_output("wfl"); // wavefield
        
    // define gpu to be used
    int gpu;
    if (! sf_getint("gpu", &gpu)) gpu = 0; //gpu id
    sf_warning("using gpu #%d", gpu);
    hipSetDevice(gpu);

    // set up axis
    at = sf_iaxa(Fwav,2); sf_setlabel(at,"t"); if(verb) sf_raxa(at); //time
    az = sf_iaxa(Fvel,1); sf_setlabel(az,"z"); if(verb) sf_raxa(az); //depth
    ay = sf_iaxa(Fvel,3); sf_setlabel(ay,"y"); if(verb) sf_raxa(ay); //y
    ax = sf_iaxa(Fvel,2); sf_setlabel(ax,"x"); if(verb) sf_raxa(ax); //x

    as = sf_iaxa(Fsou,2); sf_setlabel(as,"s"); if(verb) sf_raxa(as); //sources    
    ar = sf_iaxa(Frec,2); sf_setlabel(ar,"r"); if(verb) sf_raxa(ar); //receivers
    sf_axis ar_3, as_3;
    ar_3 = sf_iaxa(Frec, 3);
    as_3 = sf_iaxa(Fsou, 3);

    awt = at;

    nt = sf_n(at); dt = sf_d(at);
    nz = sf_n(az); dz = sf_d(az);
    ny = sf_n(ay); dy = sf_d(ay);
    nx = sf_n(ax); dx = sf_d(ax);

    ns = sf_n(as_3) * sf_n(as);
    nr = sf_n(ar_3) * sf_n(ar);

    sf_warning("nx:%d|ny:%d|nz:%d|nt:%d|ns:%d|nr:%d",nx,ny,nz,nt,ns,nr);
    sf_warning("dx:%f|dy:%f|dz:%f|dt:%f", dx, dy, dz, dt);
    
    // define bell size
    if(! sf_getint("nbell",&nbell)) nbell=5;  //bell size
    sf_warning("nbell=%d",nbell);
    
    // how often to extract receiver data?
    if(! sf_getint("jdata",&jdata)) jdata=1;
    
    if(snap) {

        if(! sf_getint("jsnap",&jsnap)) jsnap=nt;       // save wavefield every jsnap time steps 
    
        sf_warning("extracting recevier data every %d timesteps", jsnap);
        
	acz = sf_maxa(nz, sf_o(az), dz);
    acx = sf_maxa(nx, sf_o(ax), dx);
	acy = sf_maxa(ny, sf_o(ay), dy);

	int ntsnap;
        ntsnap=0;
        for(it=0; it<nt; it++) {
            if(it%jsnap==0) ntsnap++;
        }
        
	sf_warning("therefore there are %d extractions", ntsnap);
   
        sf_setn(awt,ntsnap);
        sf_setd(awt,dt*jsnap);

	sf_oaxa(Fwfl, acz, 1);
	sf_oaxa(Fwfl, acx, 2);
	sf_oaxa(Fwfl, acy, 3);

	sf_oaxa(Fwfl, awt, 4);

    }
    
    // how many time steps in each extraction?
    int nsmp = (nt/jdata);
    if(! sf_getint("jdata",&jdata)) jdata=1;    // extract receiver data every jdata time steps 

    
    // define increase in domain of model for boundary conditions
    if( !sf_getint("nb",&nb) || nb<NOP) nb=NOP;
    
    // init fdm
    fdm=fdutil3d_init(verb,fsrf,az,ax,ay,nb,1);
    sf_warning("ox %f, oy %f, oz %f", fdm->oxpad, fdm->oypad, fdm->ozpad);

    // MOVE SOURCE WAVELET INTO GPU
    // for this we basically have to compute the weights of the wavelet to make it correct when on the grid
    ncs = 1;
    float *ww=NULL;
    ww = sf_floatalloc(nt); // allocate variable for ncs dimensions over nt time
    sf_floatread(ww, nt, Fwav); // read the wavelet into the allocated memory

    float *h_ww;
    h_ww = (float*)malloc(nt*sizeof(float));
    for (int t = 0; t < nt; t++) {
        h_ww[t] = ww[t];
    }

    float *d_ww;
    hipMalloc((void**)&d_ww, 1*ncs*nt*sizeof(float));
    sf_check_gpu_error("hipMalloc source wavelet to device");
    hipMemcpy(d_ww, h_ww, 1*ncs*nt*sizeof(float), hipMemcpyHostToDevice);
    sf_check_gpu_error("copy source wavelet to device");
    
    // SETUP SOURCE/RECEIVER COORDS
    pt3d *ss=NULL;
    pt3d *rr=NULL;

    ss = (pt3d*) sf_alloc(ns, sizeof(*ss));
    rr = (pt3d*) sf_alloc(nr, sizeof(*rr));
    
    float *d_Sw000, *d_Sw001, *d_Sw010, *d_Sw011, *d_Sw100, *d_Sw101, *d_Sw110, *d_Sw111;
    hipMalloc((void**)&d_Sw000, ns * sizeof(float));
    hipMalloc((void**)&d_Sw001, ns * sizeof(float));
    hipMalloc((void**)&d_Sw010, ns * sizeof(float));
    hipMalloc((void**)&d_Sw011, ns * sizeof(float));
    hipMalloc((void**)&d_Sw100, ns * sizeof(float));
    hipMalloc((void**)&d_Sw101, ns * sizeof(float));
    hipMalloc((void**)&d_Sw110, ns * sizeof(float));
    hipMalloc((void**)&d_Sw111, ns * sizeof(float));
    sf_check_gpu_error("hipMalloc source interpolation coefficients to device");

    // z and x,y coordinates of each source
    int *d_Sjz, *d_Sjx, *d_Sjy;
    hipMalloc((void**)&d_Sjz, ns * sizeof(int));
    hipMalloc((void**)&d_Sjx, ns * sizeof(int));
    hipMalloc((void**)&d_Sjy, ns * sizeof(int));
    sf_check_gpu_error("hipMalloc source coords to device");

    float *d_Rw000, *d_Rw001, *d_Rw010, *d_Rw011, *d_Rw100, *d_Rw101, *d_Rw110, *d_Rw111;
    hipMalloc((void**)&d_Rw000, nr * sizeof(float));
    hipMalloc((void**)&d_Rw001, nr * sizeof(float));
    hipMalloc((void**)&d_Rw010, nr * sizeof(float));
    hipMalloc((void**)&d_Rw011, nr * sizeof(float));
    hipMalloc((void**)&d_Rw100, nr * sizeof(float));
    hipMalloc((void**)&d_Rw101, nr * sizeof(float));
    hipMalloc((void**)&d_Rw110, nr * sizeof(float));
    hipMalloc((void**)&d_Rw111, nr * sizeof(float));
    sf_check_gpu_error("hipMalloc receiver interpolation coefficients to device");

    // z and x coordinates of each receiver
    int *d_Rjz, *d_Rjx, *d_Rjy;
    hipMalloc((void**)&d_Rjz, nr * sizeof(int));
    hipMalloc((void**)&d_Rjx, nr * sizeof(int));
    hipMalloc((void**)&d_Rjy, nr * sizeof(int));
    sf_check_gpu_error("hipMalloc receiver coords to device");

    // read density and stiffness
    float *tt1 = (float*)malloc(nz * nx * ny * sizeof(float)); // array to transfer data with
    // allocate host stiffness (h_vel)
    h_vel=(float*)malloc(fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float));

    sf_warning("Expanding dimensions for boundary conditions to:");
    sf_warning("nxpad: %d | nypad: %d | nzpad: %d", fdm->nxpad, fdm->nypad, fdm->nzpad);
    
    // read and expand velocity
    sf_floatread(tt1,nz*nx*ny,Fvel);
    expand_cpu_3d(tt1, h_vel, fdm->nb, nx, fdm->nxpad, ny, fdm->nypad, nz, fdm->nzpad);
   
    hipMalloc((void **)&d_vel, fdm->nzpad*fdm->nxpad*fdm->nypad*sizeof(float));
    sf_check_gpu_error("allocated velocity to device");

    hipMemcpy(d_vel, h_vel, fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float), hipMemcpyHostToDevice);
    sf_check_gpu_error("copy velocity to device");
       
    // CREATE DATA ARRAYS
    float *d_dd_pp; float *h_dd_pp; 

    h_dd_pp = (float*)malloc(nsmp * nr * sizeof(float));
    hipMalloc((void**)&d_dd_pp, nsmp*nr*sizeof(float));
    
    sf_check_gpu_error("allocate data arrays");

    // allocate grid arrays on GPUs
    hipMalloc((void **)&d_ppo, fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float));
    hipMalloc((void **)&d_po, fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float));
    hipMalloc((void **)&d_fpo, fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float));
    h_po=(float*)malloc(fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float));
    sf_check_gpu_error("allocate grid arrays");

    // create array for wavefield
    if (snap) {
        oslice = sf_floatalloc3(sf_n(az), sf_n(ax), sf_n(ay));
        po = sf_floatalloc3(fdm->nzpad, fdm->nxpad, fdm->nypad);
    }

    if (bnds) {
	sf_setn(acz, fdm->nzpad);
	sf_setn(acx, fdm->nxpad);
	sf_setn(acy, fdm->nypad);

	sf_oaxa(Fwfl, acz, 1);
        sf_oaxa(Fwfl, acx, 2);
        sf_oaxa(Fwfl, acy, 3);
    }

    // SET UP ONE WAY BOUND CONDITIONS
    float *one_bzl = sf_floatalloc(fdm->nxpad * fdm->nypad);
    float *one_bzh = sf_floatalloc(fdm->nxpad * fdm->nypad);
    float *one_bxl = sf_floatalloc(fdm->nzpad * fdm->nypad);
    float *one_bxh = sf_floatalloc(fdm->nzpad * fdm->nypad);
    float *one_byl = sf_floatalloc(fdm->nxpad * fdm->nzpad);
    float *one_byh = sf_floatalloc(fdm->nxpad * fdm->nzpad);

    float d;
    for (int ix=0; ix<fdm->nxpad; ix++) {
	for (int iy=0; iy<fdm->nypad; iy++) {
            d = h_vel[iy*fdm->nxpad*fdm->nzpad + NOP*fdm->nxpad + ix] * (dt / dz);
            one_bzl[iy*fdm->nxpad+ix] = (1-d)/(1+d);
            d = h_vel[iy*fdm->nxpad*fdm->nzpad + (fdm->nzpad-NOP-1)*fdm->nxpad + ix] * (dt / dz);
            one_bzh[iy*fdm->nxpad+ix] = (1-d)/(1+d);
	}
    }
    for (int iz=0; iz<fdm->nzpad; iz++) {
	for (int iy=0; iy<fdm->nypad; iy++) {
            d = h_vel[iy*fdm->nxpad*fdm->nzpad + iz*fdm->nxpad + NOP] * (dt / dx);
            one_bxl[iy*fdm->nzpad+iz] = (1-d)/(1+d);
            d = h_vel[iy*fdm->nxpad*fdm->nzpad + iz*fdm->nxpad + fdm->nxpad-NOP-1] * (dt / dx);
            one_bxh[iy*fdm->nzpad+iz] = (1-d)/(1+d);
	}
    }
    for (int iz=0; iz<fdm->nzpad; iz++) {
	for (int ix=0; ix<fdm->nxpad; ix++) {
	    d = h_vel[NOP*fdm->nxpad*fdm->nzpad + iz*fdm->nxpad + ix] * (dt / dy);
	    one_byl[iz*fdm->nxpad+ix] = (1-d)/(1+d);
	    d = h_vel[(fdm->nypad-NOP-1)*fdm->nxpad*fdm->nzpad + iz*fdm->nxpad + ix] * (dt / dy);
	    one_byh[iz*fdm->nxpad+iz] = (1-d)/(1+d);
	}
    }

    float *d_bzl, *d_bzh, *d_bxl, *d_bxh, *d_byl, *d_byh;
    hipMalloc((void**)&d_bzl, fdm->nxpad*fdm->nypad*sizeof(float));
    hipMemcpy(d_bzl, one_bzl, fdm->nxpad*fdm->nypad*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_bzh, fdm->nxpad*fdm->nypad*sizeof(float));
    hipMemcpy(d_bzh, one_bzh, fdm->nxpad*fdm->nypad*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_bxl, fdm->nzpad*fdm->nypad*sizeof(float));
    hipMemcpy(d_bxl, one_bxl, fdm->nzpad*fdm->nypad*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_bxh, fdm->nzpad*fdm->nypad*sizeof(float));
    hipMemcpy(d_bxh, one_bxh, fdm->nzpad*fdm->nypad*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_byl, fdm->nzpad*fdm->nxpad*sizeof(float));
    hipMemcpy(d_byl, one_byl, fdm->nzpad*fdm->nxpad*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_byh, fdm->nzpad*fdm->nxpad*sizeof(float));
    hipMemcpy(d_byh, one_byh, fdm->nzpad*fdm->nxpad*sizeof(float), hipMemcpyHostToDevice);

    // ITERATE OVER SHOTS
    for (int isrc = 0; isrc < 1; isrc++){

        sf_warning("Modeling shot %d", isrc+1);

	pt3dread1(Fsou, ss, ns, 3); // read source coords
	pt3dread1(Frec, rr, nr, 3); // read receiver coords
	
	// SET SOURCE ON GPU
	sf_warning("Source location: ");
	printpt3d(*ss);

    // do 3d linear interpolation to find source location
	cs = lint3d_make(ns, ss, fdm);	

	sf_warning("Source interp coeffs:");
	sf_warning("000:%f | 001:%f | 010:%f | 011:%f | 100:%f | 101:%f | 110:%f | 111:%f", cs->w000[0], cs->w001[0], cs->w010[0], cs->w011[0], cs->w100[0], cs->w101[0], cs->w101[0], cs->w111[0]); 

	hipMemcpy(d_Sw000, cs->w000, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw001, cs->w001, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw010, cs->w010, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw011, cs->w011, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw100, cs->w100, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw101, cs->w101, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw110, cs->w110, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw111, cs->w111, ns * sizeof(float), hipMemcpyHostToDevice);
    sf_check_gpu_error("copy source interpolation coefficients to device");

    hipMemcpy(d_Sjz, cs->jz, ns * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Sjx, cs->jx, ns * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Sjy, cs->jy, ns * sizeof(int), hipMemcpyHostToDevice);
    sf_check_gpu_error("copy source coords to device");

	
	// SET RECEIVERS ON GPU
	sf_warning("Receiver Count: %d", nr);
	cr = lint3d_make(nr, rr, fdm);

    hipMemcpy(d_Rw000, cr->w000, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw001, cr->w001, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw010, cr->w010, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw011, cr->w011, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw100, cr->w100, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw101, cr->w101, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw110, cr->w110, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw111, cr->w111, nr * sizeof(float), hipMemcpyHostToDevice);
    sf_check_gpu_error("copy receiver interpolation coefficients to device");

    hipMemcpy(d_Rjz, cr->jz, nr * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Rjx, cr->jx, nr * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Rjy, cr->jy, nr * sizeof(int), hipMemcpyHostToDevice);
    sf_check_gpu_error("copy receiver coords to device");
	
	
	// set pressure to 0 on gpu
	hipMemset(d_ppo, 0, fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float));
    hipMemset(d_po, 0, fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float));
	hipMemset(d_fpo, 0, fdm->nzpad * fdm->nxpad * fdm->nypad * sizeof(float));
	sf_check_gpu_error("initialize grid arrays");

	// set data to zero
    hipMemset(d_dd_pp, 0, nsmp * nr * sizeof(float));

    for (int i = 0; i < nsmp * nr; i++){
        h_dd_pp[i] = 0.f;
    }

	// -= TIME LOOP =-
	if(verb) fprintf(stderr,"\n");
    sf_warning("total number of time steps: %d", nt);

	int itr = 0; int wfnum = 0;
	for (it=0; it<nt; it++) {
	    
	    fprintf(stderr, "\b\b\b\b\b\b\b\b\b\b\b\b\b\b\btime step: %d", it+1);
	    
	    // INJECT PRESSURE SOURCE
        dim3 dimGridS(MIN(ns, ceil(ns/1024.0f)), 1, 1);
	    dim3 dimBlockS(MIN(ns, 1024), 1, 1);
        inject_sources_3D<<<dimGridS, dimBlockS>>>(d_po, d_ww, 
			    d_Sw000, d_Sw001, d_Sw010, d_Sw011, 
			    d_Sw100, d_Sw101, d_Sw110, d_Sw111, 
			    d_Sjx, d_Sjy, d_Sjz, 
			    it, ns, fdm->nxpad, fdm->nypad, fdm->nzpad);
        sf_check_gpu_error("inject_sources Kernel");

	    dim3 dimGrid4(ceil(fdm->nxpad/8.0f),ceil(fdm->nypad/8.0f),ceil(fdm->nzpad/8.0f));
        dim3 dimBlock4(8,8,8);
	    solve_3D<<<dimGrid4, dimBlock4>>>(d_fpo, d_po, d_ppo,
                                        d_vel,
                                        dx, dy, dz, dt,
                                        fdm->nxpad, fdm->nypad, fdm->nzpad);
	    sf_check_gpu_error("solve Kernel");

	    shift_3D<<<dimGrid4, dimBlock4>>>(d_fpo, d_po, d_ppo,
			    		   fdm->nxpad, fdm->nypad, fdm->nzpad);
	    sf_check_gpu_error("shift Kernel");

	    // APPLY FREE SURFACE BOUNDARY CONDITION   
	    dim3 dimGrid3(ceil(fdm->nxpad/8.0f), ceil(fdm->nypad/8.0f), ceil(fdm->nzpad/8.0f));
	    dim3 dimBlock3(8,8,8);
	    freeSurf_3D<<<dimGrid3,dimBlock3>>>(d_po, fdm->nxpad, fdm->nypad, fdm->nzpad, fdm->nb);
	    sf_check_gpu_error("freeSurf Kernel");

	    // ONE WAY BC
	    onewayBC_3D<<<dimGrid3,dimBlock3>>>(d_po, d_ppo, 
			                     d_bzl, d_bzh, d_bxl, d_bxh, d_byl, d_byh, 
					     fdm->nxpad, fdm->nypad, fdm->nzpad);
	    
	    // APPLY SPONGE BOUNDARY CONDITION
	    spongeKernel_3D<<<dimGrid3, dimBlock3>>>(d_po, fdm->nxpad, fdm->nypad, fdm->nzpad, fdm->nb);
	    sf_check_gpu_error("sponge Kernel");
	    spongeKernel_3D<<<dimGrid3, dimBlock3>>>(d_ppo, fdm->nxpad, fdm->nypad, fdm->nzpad, fdm->nb);
        sf_check_gpu_error("sponge Kernel");
	    
	    // MOVE DATA TO GPU
	    if (it % jdata == 0) {
		itr += 1;
		dim3 dimGrid_extract(MIN(nr, ceil(nr/1024.0f)), 1, 1);
		dim3 dimBlock_extract(MIN(nr, 1024), 1, 1);
		extract_3D<<<dimGrid_extract, dimBlock_extract>>>(d_dd_pp, 
                                        itr, nr, fdm->nxpad, fdm->nypad, fdm->nzpad,
									  d_po, d_Rjx, d_Rjy, d_Rjz,
									  d_Rw000, d_Rw001, d_Rw010, d_Rw011,
									  d_Rw100, d_Rw101, d_Rw110, d_Rw111);
		sf_check_gpu_error("lint3d_extract_gpu Kernel");
	    }

	    // EXTRACT WAVEFIELD EVERY JSNAP STEPS
	    if (snap && it % jsnap == 0) {
		
            hipMemcpy(h_po, d_po, fdm->nxpad*fdm->nypad*fdm->nzpad*sizeof(float), hipMemcpyDefault);
                
            for (int x = 0; x < fdm->nxpad; x++){
                for (int z = 0; z < fdm->nzpad; z++){
                    for (int y = 0; y < fdm->nypad; y++) { 
                        po[y][x][z] = h_po[y*fdm->nzpad*fdm->nxpad + z * fdm->nxpad + x];
                    }
                }
            }
        
            if (bnds){
                sf_floatwrite(po[0][0], fdm->nzpad*fdm->nxpad*fdm->nypad, Fwfl);
            } else {	    
                cut3d(po, oslice, fdm, az, ax, ay);
                sf_floatwrite(oslice[0][0], sf_n(az)*sf_n(ax)*sf_n(ay), Fwfl);
            }

	    }
  
	}

	hipMemcpy(h_dd_pp, d_dd_pp, nsmp*nr*sizeof(float), hipMemcpyDefault);
	
	sf_setn(ar, nr);
	sf_setn(at, nsmp);
	sf_setd(at, dt*jdata);

	sf_oaxa(Fdat, at, 2);
	sf_oaxa(Fdat, ar, 1);

	sf_floatwrite(h_dd_pp, nsmp*nr, Fdat);
        
    }
}
