#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

extern "C" {
    #include <rsf.h>
    #include <rsf_su.h>
}

#include "fdutil.c"

#include "spher_kernels.cu"

#define MIN(x, y) (((x) < (y)) ? (x): (y))
#define NOP 4

// funct to check gpu error
static void sf_check_gpu_error (const char *msg) {
    hipError_t err = hipGetLastError ();
    if (hipSuccess != err)
	sf_error ("Cuda error: %s: %s", msg, hipGetErrorString (err));
}

// entry
int main(int argc, char*argv[]) {

    // define input variables from sconstruct
    bool fsrf, snap, bnds, dabc;
    int jsnap, jdata;

    // define IO files
    sf_file Fwav=NULL; //wavelet
    sf_file Fsou=NULL; //sources
    sf_file Frec=NULL; //receivers
    sf_file Fvel=NULL; //velocity

    sf_file Fdat=NULL; //data
    sf_file Fwfl=NULL;

    // define axis
    sf_axis at, awt, ara, ath, aph, acra, acth, acph; // time, radius, theta, phi
    sf_axis as, ar;

    // define dimension sizes
    int nt, nra, nth, nph, ns, nr, ncs, nb;
    int it;
    float dt, dra, dth, dph;

    // FDM structure
    fdm3d fdm=NULL;

    // device and host velocity
    float *h_vel, *d_vel;
    
    // pressure
    float *d_fpo, *d_po, *d_ppo; // future, present, past
    
    // vars for wavefield return
    float *h_po;
    float ***po=NULL;
    float ***oslice=NULL;

    // linear interpolation of weights and indicies
    lint3d cs, cr;

    sf_init(argc, argv);

    // exec flags
    if(! sf_getbool("free",&fsrf)) fsrf=false; /* free surface flag */
    if(! sf_getbool("snap",&snap)) snap=false;
    if(! sf_getbool("dabc",&dabc)) dabc=false; /* absorbing BC */
    if(! sf_getbool("bnds",&bnds)) bnds=false;
    sf_warning("Free Surface: %b", fsrf);
    sf_warning("Absorbing Boundaries: %b", dabc);
    sf_warning("Saving wavefield? %b", snap);

    // IO
    Fwav = sf_input("in");
    Fvel = sf_input("vel");
    Fsou = sf_input("sou");
    Frec = sf_input("rec");

    Fdat = sf_output("out");
    Fwfl = sf_output("wfl");

    // define gpu to be used
    int gpu;
    if (! sf_getint("gpu", &gpu)) gpu = 0;
    sf_warning("Using gpu #%d", gpu);
    hipSetDevice(gpu);

    // set up axis
    at  = sf_iaxa(Fwav,2); sf_setlabel(at ,"t" ); // time
    ara = sf_iaxa(Fvel,1); sf_setlabel(ara,"ra"); // radius
    ath = sf_iaxa(Fvel,2); sf_setlabel(ath,"th"); // theta
    aph = sf_iaxa(Fvel,3); sf_setlabel(aph,"ph"); // phi

    as  = sf_iaxa(Fsou,2); sf_setlabel(as ,"s" ); // sources
    ar  = sf_iaxa(Frec,2); sf_setlabel(ar ,"r" ); // receivers

    awt = at;

    nt  = sf_n(at ); dt  = sf_d(at );
    nra = sf_n(ara); dra = sf_d(ara);
    nth = sf_n(ath); dth = sf_d(ath);
    nph = sf_n(aph); dph = sf_d(aph);
    
    ns  = sf_n(as);
    nr  = sf_n(ar);

    sf_warning("nra:%d|nth:%d|nph:%d|nt:%d|ns:%d|nr:%d",nra,nth,nph,nt,ns,nr);
    sf_warning("dra:%f|dth:%f|dph:%f|dt:%f", dra, dth, dph, dt);

    
    // define increase in domain of model for boundary conditions
    if( !sf_getint("nb",&nb) || nb<NOP) nb=NOP;

    // init FDM
    // FDM is based on Z, X, Y. Not spherical. So we need to convert
    // to spherical. Z=Theta, X=Radius, Y=Phi
    fdm = fdutil3d_init(false, fsrf, ath, ara, aph, nb, 1);
    float oth, ora, oph;
    oth = fdm->ozpad; ora = fdm->oxpad; oph = fdm->oypad;
    sf_warning("oth %f, ora %f, oph %f", fdm->ozpad, fdm->oxpad, fdm->oypad);

    // x, y, z pad to nrapad, nthpad, nphpad
    int nrapad=fdm->nxpad; int nthpad=fdm->nzpad; int nphpad=fdm->nypad;
    sf_warning("nrapad: %d | nthpad: %d | nphpad: %d", nrapad, nthpad, nphpad);
    h_vel = (float*)malloc(nrapad * nthpad * nphpad * sizeof(float));
    
    // how often to extract receiver data?
    if(! sf_getint("jdata",&jdata)) jdata=1;
    int nsmp = (nt/jdata);
    sf_warning("reading receiver data %d times", nsmp);

    sf_warning("nb: %d", nb);
    
    if(snap) {

        if(! sf_getint("jsnap",&jsnap)) jsnap=nt; // save wavefield every nt timesteps

        acth = sf_maxa(nth, sf_o(ath), dth); sf_setlabel(acth,"lat/th (rad)");
        acra = sf_maxa(nra, sf_o(ara), dra); sf_setlabel(acra,"ra (km)"); // radius
        acph = sf_maxa(nph, sf_o(aph), dph); sf_setlabel(acph,"lon/ph (rad)"); 

        int ntsnap = 0;
        for (it=0; it<nt; it++) {
            if(it%jsnap==0) ntsnap++;
        }

        sf_warning("There are %d wavefield extractions", ntsnap);

        sf_setn(awt, ntsnap);
        sf_setd(awt, dt*jsnap);

        if (bnds) {

            sf_setn(acth, nthpad);
            sf_setn(acra, nrapad);
            sf_setn(acph, nphpad);    
            
        }

        sf_oaxa(Fwfl, acth, 2);
        sf_oaxa(Fwfl, acra, 1);
        sf_oaxa(Fwfl, acph, 3);

        sf_oaxa(Fwfl, awt, 4);

    }

    // MOVE SOURCE WAVELET INTO THE GPU
    ncs = 1;
    float *ww = NULL;
    ww = sf_floatalloc(nt); // allocate var for ncs dims over nt time
    sf_floatread(ww, nt, Fwav); // read wavelet into allocated mem

    float *h_ww;
    h_ww = (float*)malloc(nt*sizeof(float));
    for (int t = 0; t < nt; t++) {
        h_ww[t] = ww[t];
    }

    float *d_ww;
    hipMalloc((void**)&d_ww, ncs*nt*sizeof(float));
    sf_check_gpu_error("hipMalloc source wavelet to device");
    hipMemcpy(d_ww, h_ww, ncs*nt*sizeof(float), hipMemcpyHostToDevice);

    // SET UP SOURCE / RECEIVER COORDS
    pt3d *ss=NULL;
    pt3d *rr=NULL;

    ss = (pt3d*) sf_alloc(ns, sizeof(*ss));
    rr = (pt3d*) sf_alloc(nr, sizeof(*rr));

    float *d_Sw000, *d_Sw001, *d_Sw010, *d_Sw011, *d_Sw100, *d_Sw101, *d_Sw110, *d_Sw111;
    hipMalloc((void**)&d_Sw000, ns * sizeof(float));
    hipMalloc((void**)&d_Sw001, ns * sizeof(float));
    hipMalloc((void**)&d_Sw010, ns * sizeof(float));
    hipMalloc((void**)&d_Sw011, ns * sizeof(float));
    hipMalloc((void**)&d_Sw100, ns * sizeof(float));
    hipMalloc((void**)&d_Sw101, ns * sizeof(float));
    hipMalloc((void**)&d_Sw110, ns * sizeof(float));
    hipMalloc((void**)&d_Sw111, ns * sizeof(float));
    sf_check_gpu_error("hipMalloc source interpolation coefficients to device");

    // radal and theta, phi coordinates of each source
    int *d_Sjra, *d_Sjth, *d_Sjph;
    hipMalloc((void**)&d_Sjra, ns * sizeof(int));
    hipMalloc((void**)&d_Sjth, ns * sizeof(int));
    hipMalloc((void**)&d_Sjph, ns * sizeof(int));
    sf_check_gpu_error("hipMalloc source coords to device");

    float *d_Rw000, *d_Rw001, *d_Rw010, *d_Rw011, *d_Rw100, *d_Rw101, *d_Rw110, *d_Rw111;
    hipMalloc((void**)&d_Rw000, nr * sizeof(float));
    hipMalloc((void**)&d_Rw001, nr * sizeof(float));
    hipMalloc((void**)&d_Rw010, nr * sizeof(float));
    hipMalloc((void**)&d_Rw011, nr * sizeof(float));
    hipMalloc((void**)&d_Rw100, nr * sizeof(float));
    hipMalloc((void**)&d_Rw101, nr * sizeof(float));
    hipMalloc((void**)&d_Rw110, nr * sizeof(float));
    hipMalloc((void**)&d_Rw111, nr * sizeof(float));
    sf_check_gpu_error("hipMalloc receiver interpolation coefficients to device");

    // radial, theta, and phi locations of each receiver
    int *d_Rjra, *d_Rjth, *d_Rjph;
    hipMalloc((void**)&d_Rjra, nr * sizeof(int));
    hipMalloc((void**)&d_Rjth, nr * sizeof(int));
    hipMalloc((void**)&d_Rjph, nr * sizeof(int));
    sf_check_gpu_error("hipMalloc receiver coords to device");

    // allocate memory to import velocity data
    float *tt1 = (float*)malloc(nra * nth * nph * sizeof(float));
    
    // read in velocity data & expand domain
    sf_floatread(tt1, nra*nth*nph, Fvel);
    expand_cpu_3D(tt1, h_vel, fdm->nb, nra, nrapad, nph, nphpad, nth, nthpad);
    hipMalloc((void **)&d_vel, nthpad*nrapad*nphpad*sizeof(float));
    sf_check_gpu_error("allocated velocity to device");
    hipMemcpy(d_vel, h_vel, nthpad*nrapad*nphpad*sizeof(float), hipMemcpyHostToDevice);
    sf_check_gpu_error("copy velocity to device");

    // CREATE DATA ARRAYS FOR RECEIVERS
    float *d_dd_pp; float *h_dd_pp;
    h_dd_pp = (float*)malloc(nsmp * nr * sizeof(float));
    hipMalloc((void**)&d_dd_pp, nsmp * nr * sizeof(float));
    sf_check_gpu_error("allocate data arrays");

    // allocate pressure arrays for past, present and future on GPU's
    hipMalloc((void**)&d_ppo, nthpad*nphpad*nrapad*sizeof(float));
    hipMalloc((void**)&d_po , nthpad*nphpad*nrapad*sizeof(float));
    hipMalloc((void**)&d_fpo, nthpad*nphpad*nrapad*sizeof(float));
    h_po=(float*)malloc(nthpad * nrapad * nphpad * sizeof(float));
    sf_check_gpu_error("allocate pressure arrays");
    
    if (snap) {

        oslice = sf_floatalloc3(sf_n(ara), sf_n(ath), sf_n(aph));
        po = sf_floatalloc3(nrapad, nthpad, nphpad);
    
    }
    
    // SET UP ONE WAY BOUND CONDITIONS
    float *one_bthl = sf_floatalloc(nrapad * nphpad);
    float *one_bthh = sf_floatalloc(nrapad * nphpad);
    float *one_bral = sf_floatalloc(nthpad * nphpad);
    float *one_brah = sf_floatalloc(nthpad * nphpad);
    float *one_bphl = sf_floatalloc(nrapad * nthpad);
    float *one_bphh = sf_floatalloc(nrapad * nthpad);

    float d;
    for (int ira=0; ira<nrapad; ira++) {
        for (int iph=0; iph<nphpad; iph++) {
            d = h_vel[iph*nrapad*nthpad + NOP*nrapad + ira] * (dt / dth);
            one_bthl[iph*nrapad+ira] = (1-d)/(1+d);
            d = h_vel[iph*nrapad*nthpad + (nthpad-NOP-1)*nrapad + ira] * (dt / dth);
            one_bthh[iph*nrapad+ira] = (1-d)/(1+d);
        }
    }
    for (int ith=0; ith<nthpad; ith++) {
        for (int iph=0; iph<nphpad; iph++) {
            d = h_vel[iph*nrapad*nthpad + ith*nrapad + NOP] * (dt / dra);
            one_bral[iph*nthpad+ith] = (1-d)/(1+d);
            d = h_vel[iph*nrapad*nthpad + ith*nrapad + nrapad-NOP-1] * (dt / dra);
            one_brah[iph*nthpad+ith] = (1-d)/(1+d);
        }
    }
    for (int ith=0; ith<nthpad; ith++) {
        for (int ira=0; ira<nrapad; ira++) {
            d = h_vel[NOP*nrapad*nthpad + ith*nrapad + ira] * (dt / dph);
            one_bphl[ith*nrapad+ira] = (1-d)/(1+d);
            d = h_vel[(nthpad-NOP-1)*nrapad*nthpad + ith*nrapad + ira] * (dt / dph);
            one_bphh[ith*nrapad+ira] = (1-d)/(1+d);
        }
    }
    
    float *d_bthl, *d_bthh, *d_bral, *d_brah, *d_bphl, *d_bphh;
    hipMalloc((void**)&d_bthl, nrapad*nphpad*sizeof(float));
    hipMemcpy(d_bthl, one_bthl, nrapad*nphpad*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_bthh, nrapad*nphpad*sizeof(float));
    hipMemcpy(d_bthh, one_bthh, nrapad*nphpad*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_bral, nthpad*nphpad*sizeof(float));
    hipMemcpy(d_bral, one_bral, nthpad*nphpad*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_brah, nthpad*nphpad*sizeof(float));
    hipMemcpy(d_brah, one_brah, nthpad*nphpad*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_bphl, nthpad*nrapad*sizeof(float));
    hipMemcpy(d_bphl, one_bphl, nthpad*nrapad*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_bphh, nthpad*nrapad*sizeof(float));
    hipMemcpy(d_bphh, one_bphh, nthpad*nrapad*sizeof(float), hipMemcpyHostToDevice);
    sf_check_gpu_error("copy one way bc conditions to device");

    // ITERATE OVER SHOTS
    // CURRENTLY ONLY 1 TO SET ALL SHOTS AT ONCE
    for (int isrc = 0; isrc < 1; isrc ++) {

	// read source and receiver coordinates
	// in the pt3d struct there is X, Y and Z. The same convention is
	// used here to transform into spherical coordinates (X:Radius,
	// Y:Phi, Z:Theta)
	pt3dread1(Fsou, ss, ns, 3);
	pt3dread1(Frec, rr, nr, 3);

	// SET SOURCES ON GPU

	// perform 3d linear interpolation on source
	cs = lint3d_make(ns, ss, fdm);

    hipMemcpy(d_Sw000, cs->w000, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw001, cs->w001, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw010, cs->w010, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw011, cs->w011, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw100, cs->w100, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw101, cs->w101, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw110, cs->w110, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw111, cs->w111, ns * sizeof(float), hipMemcpyHostToDevice);
    sf_check_gpu_error("copy source interpolation coefficients to device");

    hipMemcpy(d_Sjth, cs->jz, ns * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_Sjra, cs->jx, ns * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_Sjph, cs->jy, ns * sizeof(int), hipMemcpyHostToDevice);
	sf_check_gpu_error("copy source coords to device");

	// SET RECEIVERS ON THE GPU
	cr = lint3d_make(nr, rr, fdm);

	hipMemcpy(d_Rw000, cr->w000, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw001, cr->w001, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw010, cr->w010, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw011, cr->w011, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw100, cr->w100, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw101, cr->w101, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw110, cr->w110, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw111, cr->w111, nr * sizeof(float), hipMemcpyHostToDevice);
    sf_check_gpu_error("copy receiver interpolation coefficients to device");

    hipMemcpy(d_Rjth, cr->jz, nr * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Rjra, cr->jx, nr * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Rjph, cr->jy, nr * sizeof(int), hipMemcpyHostToDevice);
    sf_check_gpu_error("copy receiver coords to device");


	// set pressure to 0 on gpu
	hipMemset(d_ppo, 0, nthpad*nphpad*nrapad*sizeof(float));
	hipMemset(d_po , 0, nthpad*nphpad*nrapad*sizeof(float));
	hipMemset(d_fpo, 0, nthpad*nphpad*nrapad*sizeof(float));
	sf_check_gpu_error("Set pressure arrays to 0");

	// set receiver data to 0
	hipMemset(d_dd_pp, 0, nsmp*nr*sizeof(float));

	for (int i=0; i < nsmp*nr; i++) {
	    h_dd_pp[i] = 0.f;
	}

	// TIME LOOP
	fprintf(stderr,"total num of time steps: %d \n", nt);
	for (it=0; it<nt; it++) {

	    fprintf(stderr, "\b\b\b\b\b\b\b\b\b\b\b\b\b\b\btime step: %d", it+1);

	    // INJECT PRESSURE SOURCE
        
        dim3 dimGridS(MIN(ns, ceil(ns/1024.0f)), 1, 1);
	    dim3 dimBlockS(MIN(ns, 1024), 1, 1);
        inject_sources_3D<<<dimGridS, dimBlockS>>>(d_po, d_ww, 
			    d_Sw000, d_Sw001, d_Sw010, d_Sw011, 
			    d_Sw100, d_Sw101, d_Sw110, d_Sw111, 
			    d_Sjra, d_Sjph, d_Sjth, 
			    it, ns, nrapad, nphpad, nthpad);
        sf_check_gpu_error("inject_sources Kernel");

	    dim3 dimGrid2(ceil(nrapad/8.0f),ceil(nphpad/8.0f),ceil(nthpad/8.0f));
	    dim3 dimBlock2(8,8,8);
        
	    // APPLY WAVE EQUATION
	    solve_3D<<<dimGrid2, dimBlock2>>>(d_fpo, d_po, d_ppo,
                d_vel,
                dra, dph, dth, ora, oph, oth, dt,
                nrapad, nphpad, nthpad);
	    sf_check_gpu_error("solve Kernel");

	    // SHIFT PRESSURE FIELDS IN TIME
	    shift_3D<<<dimGrid2, dimBlock2>>>(d_fpo, d_po, d_ppo,
                nrapad, nphpad, nthpad);
	    sf_check_gpu_error("shift Kernel");

	    // ONE WAY BC
	    onewayBC_3D<<<dimGrid2,dimBlock2>>>(d_po, d_ppo,
                d_bthl, d_bthh, d_bral, d_brah, d_bphl, d_bphh,
                nrapad, nphpad, nthpad);

	    // SPONGE
	    spongeKernel_3D<<<dimGrid2, dimBlock2>>>(d_po, nrapad, nphpad, nthpad, nb);
	    sf_check_gpu_error("sponge Kernel1");
	    spongeKernel_3D<<<dimGrid2, dimBlock2>>>(d_ppo, nrapad, nphpad, nthpad, nb);
        sf_check_gpu_error("sponge Kernel2");

	    // FREE SURFACE
        if (fsrf) {
            freeSurf_3D<<<dimGrid2, dimBlock2>>>(d_po, nrapad, nphpad, nthpad, nb);
            sf_check_gpu_error("free surface Kernel");
        }
		
	    // RECEIVERS
	    dim3 dimGridR(MIN(nr, ceil(nr/1024.0f)), 1, 1);
	    dim3 dimBlockR(MIN(nr, 1024), 1, 1);
	    extract_3D<<<dimGridR, dimBlockR>>>(d_dd_pp, it, nr,
                nrapad, nphpad, nthpad, 
                d_po, d_Rjra, d_Rjph, d_Rjth,
                d_Rw000, d_Rw001, d_Rw010, d_Rw011,
                d_Rw100, d_Rw101, d_Rw110, d_Rw111);
        sf_check_gpu_error("lint3d_extract_gpu Kernel");

	    // EXTRACT WAVEFIELD EVERY JSNAP STEPS
	    if (snap && it % jsnap == 0) {

            hipMemcpy(h_po, d_po, nrapad * nphpad * nthpad * sizeof(float), hipMemcpyDefault);

            if (bnds) {
                sf_floatwrite(h_po, nthpad*nrapad*nphpad, Fwfl);
            } else {
                cut3d(po, oslice, fdm, ath, ara, aph);
                sf_floatwrite(oslice[0][0], sf_n(ath)*sf_n(ara)*sf_n(aph), Fwfl);
            }
            
	    }	    

	}

    }

    fprintf(stderr,"\n");

    hipMemcpy(h_dd_pp, d_dd_pp, nsmp*nr*sizeof(float), hipMemcpyDefault);

    sf_setn(ar, nr);
    sf_setn(at, nsmp);
    sf_setd(at, dt*jdata);

    sf_oaxa(Fdat, at, 2);
    sf_oaxa(Fdat, ar, 1);

    sf_floatwrite(h_dd_pp, nsmp*nr, Fdat);

    // FREE GPU MEMORY

    hipFree(d_ww);

    hipFree(d_Sw000); hipFree(d_Sw001); hipFree(d_Sw010); hipFree(d_Sw011);
    hipFree(d_Sw100); hipFree(d_Sw101); hipFree(d_Sw110); hipFree(d_Sw111);
    hipFree(d_Sjra); hipFree(d_Sjth); hipFree(d_Sjph);
    
    hipFree(d_Rw000); hipFree(d_Rw001); hipFree(d_Rw010); hipFree(d_Rw011);
    hipFree(d_Rw100); hipFree(d_Rw101); hipFree(d_Rw110); hipFree(d_Rw111);
    hipFree(d_Rjra); hipFree(d_Rjth); hipFree(d_Rjph);

    hipFree(d_dd_pp);
    hipFree(d_ppo); hipFree(d_po); hipFree(d_fpo);

}

