#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

extern "C" {
    #include <rsf.h>
    #include <rsf_su.h>
}

#include "fdutil.c"

#include "spher_kernels.cu"

#define MIN(x, y) (((x) < (y)) ? (x): (y))
#define NOP 4

// funct to check gpu error
static void sf_check_gpu_error (const char *msg) {
    hipError_t err = hipGetLastError ();
    if (hipSuccess != err)
	sf_error ("Cuda error: %s: %s", msg, hipGetErrorString (err));
}

// entry
int main(int argc, char*argv[]) {

    // define input variables from sconstruct
    bool fsrf, snap, bnds, dabc;
    int jsnap, jdata;

    // define IO files
    sf_file Fwav=NULL; //wavelet
    sf_file Fsou=NULL; //source
    sf_file Frec=NULL; //receivers
    sf_file Fvel=NULL; //velocity
    sf_file Fdat=NULL; //data
    sf_file Fwfl=NULL;

    // define axis
    sf_axis at, ara, ath, acra, acth; // time, radius, theta
    sf_axis as, ar;

    // define dimension sizes
    int nt, nra, nth, ns, nr, ncs, nb;
    int it;
    float dt, dra, dth;
    float ot, ora, oth;

    // FDM structure
    fdm2d fdm=NULL;

    // device and host velocity
    float *h_vel, *d_vel;
    // pressure
    float *h_po, *d_fpo, *d_po, *d_ppo; // future, present, past
    
    float **oslice=NULL;
    float **po=NULL;

    // linear interpolation of weights and indicies
    lint2d cs, cr;

    sf_init(argc, argv);

    // exec flags
    if(! sf_getbool("free",&fsrf)) fsrf=false; /* free surface flag */
    if(! sf_getbool("dabc",&dabc)) dabc=false; /* absorbing BC */
    if(! sf_getbool("snap",&snap)) snap=true;
    if(! sf_getbool("bnds",&bnds)) bnds=false;
    sf_warning("Free Surface: %b", fsrf);
    sf_warning("Absorbing Boundaries: %b", dabc);

    // IO
    Fwav = sf_input("in");
    Fvel = sf_input("vel");
    Fsou = sf_input("sou");
    Frec = sf_input("rec");
    Fdat = sf_output("out");
    Fwfl = sf_output("wfl");

    // define gpu to be used
    int gpu;
    if (! sf_getint("gpu", &gpu)) gpu = 0;
    sf_warning("Using gpu #%d", gpu);
    hipSetDevice(gpu);

    // set up axis
    at  = sf_iaxa(Fwav,2); sf_setlabel(at ,"t" ); // time
    ara = sf_iaxa(Fvel,1); sf_setlabel(ara,"ra"); // radius
    ath = sf_iaxa(Fvel,2); sf_setlabel(ath,"th"); // theta

    as  = sf_iaxa(Fsou,2); sf_setlabel(as ,"s" ); // sources
    ar  = sf_iaxa(Frec,2); sf_setlabel(ar ,"r" ); // receivers

    sf_axis ar_3, as_3;
    ar_3 = sf_iaxa(Frec, 3);
    as_3 = sf_iaxa(Fsou, 3);

    nt  = sf_n(at ); dt  = sf_d(at );
    nra = sf_n(ara); dra = sf_d(ara);
    nth = sf_n(ath); dth = sf_d(ath);
    
    ns  = sf_n(as_3) * sf_n(as);
    nr  = sf_n(ar_3) * sf_n(ar);

    ora = sf_o(ara); oth = sf_o(ath); ot = sf_o(at);

    sf_warning("nra:%d|nth:%d|nt:%d|ns:%d|nr:%d",nra,nth,nt,ns,nr);
    sf_warning("dra:%f|dth:%f|dt:%f", dra, dth, dt);
    sf_warning("ora:%f|oth:%f|ot:%f", ora, oth, ot);

    // how often to extract receiver data?
    if(! sf_getint("jdata",&jdata)) jdata=1;

    // how many time steps in each extraction?
    int nsmp = (nt/jdata);
    sf_warning("reading receiver data %d times", nsmp);

    if(! sf_getint("jdata",&jdata)) jdata=1;    // extract receiver data every jdata time steps

    // define increase in domain of model for boundary conditions
    if( !sf_getint("nb",&nb) || nb<NOP) nb=NOP;
    
    if(snap) {

        if(! sf_getint("jsnap",&jsnap)) jsnap=nt;       // save wavefield every jsnap time steps

	    sf_warning("Jsnap: %d", jsnap);
        acth = sf_maxa(nth,oth,dth); 
        acra = sf_maxa(nra,ora,dra); 
    
        int ntsnap = 0;
	    for (it=0; it<nt; it++) {
	        if (it%jsnap==0) ntsnap++;
	    }

	    sf_setn(at,ntsnap);
        sf_setd(at,dt*jsnap);
        
        sf_oaxa(Fwfl,acth,1);
        sf_oaxa(Fwfl,acra,2);
        sf_oaxa(Fwfl,at,3);

    }
    

    // init FDM
    // FDM is based on Z, X, Not polar. So we need to convert
    // to spherical. Z=Theta, X=Radius
    fdm = fdutil_init(false, fsrf, ath, ara, nb, 1);
    // origin is very slighly different under FDM due to gridsize.
    sf_warning("Adjusted Origins: oth %f, ora %f", fdm->ozpad, fdm->oxpad);
    oth = fdm->ozpad; ora = fdm->oxpad;

    // MOVE SOURCE WAVELET INTO THE GPU
    ncs = 1;
    float *ww = NULL;
    ww = sf_floatalloc(nt); // allocate var for ncs dims over nt time
    sf_floatread(ww, nt, Fwav); // read wavelet into allocated mem

    float *h_ww;
    h_ww = (float*)malloc(1 * ncs * nt*sizeof(float));
    for (int t = 0; t < nt; t++) { 
        h_ww[t] = ww[t];
    }

    float *d_ww;
    hipMalloc((void**)&d_ww, 1*ncs*nt*sizeof(float));
    sf_check_gpu_error("hipMalloc source wavelet to device");
    hipMemcpy(d_ww, h_ww, 1*ncs*nt*sizeof(float), hipMemcpyHostToDevice);

    // SET UP SOURCE / RECEIVER COORDS
    pt2d *ss=NULL;
    pt2d *rr=NULL;

    ss = (pt2d*) sf_alloc(ns, sizeof(*ss));
    rr = (pt2d*) sf_alloc(nr, sizeof(*rr));

    float *d_Sw00, *d_Sw01, *d_Sw10, *d_Sw11;
    hipMalloc((void**)&d_Sw00, ns * sizeof(float));
    hipMalloc((void**)&d_Sw01, ns * sizeof(float));
    hipMalloc((void**)&d_Sw10, ns * sizeof(float));
    hipMalloc((void**)&d_Sw11, ns * sizeof(float));
    sf_check_gpu_error("hipMalloc source interpolation coefficients to device");

    // radal and theta, phi coordinates of each source
    int *d_Sjra, *d_Sjth;
    hipMalloc((void**)&d_Sjra, ns * sizeof(int));
    hipMalloc((void**)&d_Sjth, ns * sizeof(int));
    sf_check_gpu_error("hipMalloc source coords to device");

    float *d_Rw00, *d_Rw01, *d_Rw10, *d_Rw11;
    hipMalloc((void**)&d_Rw00, nr * sizeof(float));
    hipMalloc((void**)&d_Rw01, nr * sizeof(float));
    hipMalloc((void**)&d_Rw10, nr * sizeof(float));
    hipMalloc((void**)&d_Rw11, nr * sizeof(float));
    sf_check_gpu_error("hipMalloc receiver interpolation coefficients to device");

    // radial, theta, and phi locations of each receiver
    int *d_Rjra, *d_Rjth;
    hipMalloc((void**)&d_Rjra, nr * sizeof(int));
    hipMalloc((void**)&d_Rjth, nr * sizeof(int));
    sf_check_gpu_error("hipMalloc receiver coords to device");

    // allocate memory to import velocity data
    float *tt1 = (float*)malloc(nra * nth * sizeof(float));

    // x, y, z pad to nrapad, nthpad, nphpad
    int nrapad=fdm->nxpad; int nthpad=fdm->nzpad; 
    h_vel = (float*)malloc(nrapad * nthpad * sizeof(float));

    // expand dimensions to allow for absorbing boundary conditions
    sf_warning("Expanding dimensions to allocate for bound. conditions");
    sf_warning("nrapad: %d | nthpad: %d", nrapad, nthpad);
    
    // read in velocity data & expand domain
    sf_floatread(tt1, nra*nth, Fvel);
    expand_cpu_2d(tt1, h_vel, fdm->nb, nra, nrapad, nth, nthpad);
    hipMalloc((void **)&d_vel, nthpad*nrapad*sizeof(float));
    sf_check_gpu_error("allocated velocity to device");
    hipMemcpy(d_vel, h_vel, nthpad*nrapad*sizeof(float), hipMemcpyHostToDevice);
    sf_check_gpu_error("copy velocity to device");

    // CREATE DATA ARRAYS FOR RECEIVERS
    float *d_dd_pp; float *h_dd_pp;
    h_dd_pp = (float*)malloc(nsmp * nr * sizeof(float));
    hipMalloc((void**)&d_dd_pp, nsmp*nr*sizeof(float));
    sf_check_gpu_error("allocate data arrays");

    // allocate pressure arrays for past, present and future on GPU's
    hipMalloc((void**)&d_ppo, nthpad*nrapad*sizeof(float));
    hipMalloc((void**)&d_po , nthpad*nrapad*sizeof(float));
    hipMalloc((void**)&d_fpo, nthpad*nrapad*sizeof(float));
    h_po = (float*)malloc(nthpad*nrapad*sizeof(float));
    sf_check_gpu_error("allocate pressure arrays");
 
    if (snap){
        oslice = sf_floatalloc2(nth,nra);
        po = sf_floatalloc2(nthpad,nrapad);
    }

    if (bnds){
        sf_setn(acth, nthpad);
        sf_setn(acra, nrapad);
        sf_oaxa(Fwfl,acth,1);
        sf_oaxa(Fwfl,acra,2);
    }

    // SET UP ONE WAY BC's
    float *one_bthl = sf_floatalloc(nrapad);
    float *one_bthh = sf_floatalloc(nrapad);
    float *one_bral = sf_floatalloc(nthpad);
    float *one_brah = sf_floatalloc(nthpad);

    float d;
    for (int ira=0; ira<nrapad; ira++) {
        d = h_vel[NOP * nrapad + ira] * (dt / dth);
        one_bthl[ira] = (1-d)/(1+d);
        d = h_vel[(nthpad-NOP-1)*nrapad + ira] * (dt / dth);
        one_bthh[ira] = (1-d)/(1+d);
    }
    for (int ith=0; ith<nthpad; ith++) {
        d = h_vel[ith * nrapad + NOP] * (dt / dra);
        one_bral[ith] = (1-d)/(1+d);
        d = h_vel[ith * nrapad + nrapad-NOP-1] * (dt / dra);
        one_brah[ith] = (1-d)/(1+d);
    }

    float *d_bthl, *d_bthh, *d_bral, *d_brah;
    hipMalloc((void**)&d_bthl, nrapad*sizeof(float));
    hipMemcpy(d_bthl, one_bthl, nrapad*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_bthh, nrapad*sizeof(float));
    hipMemcpy(d_bthh, one_bthh, nrapad*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_bral, nthpad*sizeof(float));
    hipMemcpy(d_bral, one_bral, nthpad*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_brah, nthpad*sizeof(float));
    hipMemcpy(d_brah, one_brah, nthpad*sizeof(float), hipMemcpyHostToDevice);

    // ITERATE OVER SHOTS
    for (int isrc = 0; isrc < 1; isrc ++) {

	// read source and receiver coordinates
	// in the pt struct there is X and Z. The same convention is
	// used here to transform into spherical coordinates (X:Radius,
	// Z:Theta)
	pt2dread1(Fsou, ss, ns , 2);
	pt2dread1(Frec, rr, nr, 2);

	// set source on GPU
	sf_warning("Source location: ");
	printpt2d(*ss);

	// perform 3d linear interpolation on source
	cs = lint2d_make(ns, ss, fdm);

	sf_warning("Source interp coeffs:");
    sf_warning("00:%f | 01:%f | 10:%f | 11:%f", cs->w00[0], cs->w01[0], cs->w10[0], cs->w11[0]);

    hipMemcpy(d_Sw00, cs->w00, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw01, cs->w01, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw10, cs->w10, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw11, cs->w11, ns * sizeof(float), hipMemcpyHostToDevice);
    sf_check_gpu_error("copy source interpolation coefficients to device");

    hipMemcpy(d_Sjth, cs->jz, ns * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_Sjra, cs->jx, ns * sizeof(int), hipMemcpyHostToDevice);
	sf_check_gpu_error("copy source coords to device");


	// SET RECEIVERS ON THE GPU
	sf_warning("Receiver Count: %d", nr);
	cr = lint2d_make(nr, rr, fdm);

	hipMemcpy(d_Rw00, cr->w00, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw01, cr->w01, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw10, cr->w10, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw11, cr->w11, nr * sizeof(float), hipMemcpyHostToDevice);
    sf_check_gpu_error("copy receiver interpolation coefficients to device");

    hipMemcpy(d_Rjth, cr->jz, nr * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Rjra, cr->jx, nr * sizeof(int), hipMemcpyHostToDevice);
    sf_check_gpu_error("copy receiver coords to device");


	// set pressure to 0 on gpu
	hipMemset(d_ppo, 0, nthpad*nrapad*sizeof(float));
	hipMemset(d_po , 0, nthpad*nrapad*sizeof(float));
	hipMemset(d_fpo, 0, nthpad*nrapad*sizeof(float));  
	sf_check_gpu_error("Set pressure arrays to 0");

	// set receiver data to 0
	hipMemset(d_dd_pp, 0, nsmp*nr*sizeof(float));

	for (int i=0; i < nsmp*nr; i++) {
	    h_dd_pp[i] = 0.f;
	}

	// TIME LOOP
	fprintf(stderr,"total num of time steps: %d \n", nt);
	for (it=0; it<nt; it++) {

	    fprintf(stderr, "\b\b\b\b\b\b\b\b\b\b\b\b\b\b\btime step: %d", it+1);

	    // INJECT PRESSURE SOURCE
        dim3 dimGridS(MIN(ns, ceil(ns/1024.0f)), 1);
        dim3 dimBlockS(MIN(ns, 1024), 1);
        inject_sources_2D<<<dimGridS,dimBlockS>>>(d_po, d_ww, 
                       d_Sw00, d_Sw01, d_Sw10, d_Sw11,
                       d_Sjra, d_Sjth, it, ns, nrapad, nthpad);
        sf_check_gpu_error("inject sources Kernel");

	    // APPLY WAVE EQUATION
	    dim3 dimGrid2(ceil(nrapad/8.0f),ceil(nthpad/8.0f));
	    dim3 dimBlock2(8,8);
	    solve_2D<<<dimGrid2, dimBlock2>>>(d_fpo, d_po, d_ppo,
			    		  d_vel,
					  dra, dth, ora, oth, dt,
					  nrapad, nthpad);
	    sf_check_gpu_error("solve Kernel");
	    
	    // SHIFT PRESSURE FIELDS IN TIME
	    shift_2D<<<dimGrid2, dimBlock2>>>(d_fpo, d_po, d_ppo,
					   nrapad, nthpad);
	    sf_check_gpu_error("shift Kernel");

	    // ONE WAY BC
	    onewayBC_2D<<<dimGrid2, dimBlock2>>>(d_po, d_ppo,
			                      d_bthl, d_bthh, d_bral, d_brah,
					      nrapad, nthpad);
	    
	    // SPONGE
	    spongeKernel_2D<<<dimGrid2, dimBlock2>>>(d_po, nrapad, nthpad, nb);
	    sf_check_gpu_error("sponge Kernel");
	    spongeKernel_2D<<<dimGrid2, dimBlock2>>>(d_ppo, nrapad, nthpad, nb);
        sf_check_gpu_error("sponge Kernel");

	    // FREE SURFACE
        if (fsrf) {
            freeSurf_2D<<<dimGrid2, dimBlock2>>>(d_po, nrapad, nthpad, nb);
            sf_check_gpu_error("freeSurf Kernel");
        }

	    if (snap && it%jsnap==0) {

            hipMemcpy(h_po, d_po, nrapad*nthpad*sizeof(float), hipMemcpyDefault);

            for (int ra = 0; ra < nrapad; ra++) {
                for (int th = 0; th < nthpad; th++) {
                    po[ra][th] = h_po[th*nrapad + ra];
                }
            }	

            if (bnds) {
                sf_floatwrite(po[0], nthpad*nrapad, Fwfl);
            }
            else {
                cut2d(po, oslice, fdm, ath, ara);
                sf_floatwrite(oslice[0], sf_n(ath)*sf_n(ara), Fwfl);
            }
	    }
	    
	    // EXTRACT TO RECEIVERS
	    dim3 dimGrid3(MIN(nr, ceil(nr/1024.0f)), 1);
	    dim3 dimBlock3(MIN(nr, 1024), 1);

	    extract_2D<<<dimGrid3, dimBlock3>>>(d_dd_pp, it, nr,
			    		     nrapad, nthpad, 
					     d_po, d_Rjra, d_Rjth,
					     d_Rw00, d_Rw01, d_Rw10, d_Rw11);
	    sf_check_gpu_error("extract Kernel");

	}

    }

    fprintf(stderr,"\n");

  
    hipMemcpy(h_dd_pp, d_dd_pp, nsmp*nr*sizeof(float), hipMemcpyDefault);

    sf_setn(ar, nr);
    sf_setn(at, nsmp);
    sf_setd(at, dt*jdata);

    sf_oaxa(Fdat, at, 2);
    sf_oaxa(Fdat, ar, 1);

    sf_floatwrite(h_dd_pp, nsmp*nr, Fdat);    

    // FREE ALLOCATED MEMORY
    hipFree(d_ww);

    hipFree(d_Sw00); hipFree(d_Sw01); hipFree(d_Sw10); hipFree(d_Sw11);
    hipFree(d_Sjra); hipFree(d_Sjth);
    
    hipFree(d_Rw00); hipFree(d_Rw01); hipFree(d_Rw10); hipFree(d_Rw11);
    hipFree(d_Rjra); hipFree(d_Rjth);

    hipFree(d_dd_pp);
    hipFree(d_ppo); hipFree(d_po); hipFree(d_fpo);

}

