
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

// macro for 1d index to simulate a 3d matrix
#define INDEX2D(ix, iz, nx) ((ix)+(iz)*(nx))
// PLACEHOLDER
#define INDEX3D(ix, iy, iz, nx, nz) ((ix)+(iz)*(nx)+(iy)*(nx)*(nz))

void expand_cpu_2d(float *a, float *b, int nb, int x_a, int x_b, int z_a, int z_b){

        // copy into other array
	for (int ix = 0; ix < x_a; ix++) {
                for (int iz = 0; iz < z_a; iz++) {
                        b[INDEX2D(ix+nb,iz+nb,x_b)] = a[INDEX2D(ix,iz,x_a)];
		}
        }
	
        // expand z direction
	for (int ix = 0; ix < x_b; ix++) {
                for (int iz = 0; iz < nb; iz++) {
                        b[INDEX2D(ix,iz,x_b)] = b[INDEX2D(ix,nb,x_b)];
                        b[INDEX2D(ix,z_b-iz-1,x_b)] = b[INDEX2D(ix,z_b-nb-1,x_b)];
                }
        }
	
        //expand x direction 
	for (int ix = 0; ix < nb; ix++) {
                for (int iz = 0; iz < z_b; iz++) {
                        b[INDEX2D(ix,iz,x_b)] = b[INDEX2D(nb,iz,x_b)];
                        b[INDEX2D(x_b-ix-1,iz,x_b)] = b[INDEX2D(x_b-nb-1,iz,x_b)];
                }
        }
	
}

__global__ void lint2d_bell_gpu(float *d_uu, float *d_ww, float *d_Sw00, float *d_Sw01, float *d_Sw10, float *d_Sw11, float *d_bell, int *d_jx, int *d_jz, int it, int nc, int ns, int c, int nbell, int nxpad) {

        int ix = threadIdx.x;
        int iz = threadIdx.y;
        int ia = blockIdx.x;

        float wa = d_ww[it * nc * ns + c * ns + ia] * d_bell[(iz * (2*nbell+1)) + ix];

	int z_comp = (d_jz[ia] - nbell) + iz;
	int x_comp = (d_jx[ia] - nbell) + ix;

        atomicAdd(&d_uu[(z_comp)     * nxpad + (x_comp    )], ((wa * d_Sw00[ia])));
        atomicAdd(&d_uu[(z_comp + 1) * nxpad + (x_comp    )], ((wa * d_Sw01[ia])));
        atomicAdd(&d_uu[(z_comp)     * nxpad + (x_comp + 1)], ((wa * d_Sw10[ia])));
        atomicAdd(&d_uu[(z_comp + 1) * nxpad + (x_comp + 1)], ((wa * d_Sw11[ia])));

}


// divergence 3d for cpml
#define NOP 4 // half of the order in space

__global__ void solve(float *d_fpo, float *d_po, float *d_ppo, float *d_vel,
                      float dx, float dz, float dt,
                      int nxpad, int nzpad) {

        int ix = threadIdx.x + blockIdx.x * blockDim.x;
        int iz = threadIdx.y + blockIdx.y * blockDim.y;	

        if (ix < nxpad && iz < nzpad){

                int globalAddr = iz * nxpad + ix;
                float pxx, pzz;
                float laplace;

                // perform only in boundaries:
                if (ix >= NOP && ix < nxpad-NOP && iz >= NOP && iz < nzpad - NOP) {
                        pxx = 0.; pzz = 0.;

                        // calculate laplacian via finite differences
                        pxx =    d_po[ix+1 + iz*nxpad] \
                              -2*d_po[ix   + iz*nxpad] \
                                +d_po[ix-1 + iz*nxpad];
                        pxx = pxx / (dx * dx);

                        pzz =    d_po[ix + (iz+1)*nxpad] \
                              -2*d_po[ix +     iz*nxpad] \
                                +d_po[ix + (iz-1)*nxpad];
                        pzz = pzz / (dz * dz);

                        laplace = pxx + pzz;

                } else {
                        laplace = 0.;
                }

                // compute pressure at next time step
                d_fpo[globalAddr] = (dt*dt)*(d_vel[globalAddr]*d_vel[globalAddr]*laplace) + 2*d_po[globalAddr] - d_ppo[globalAddr];

        }

}


__global__ void shift(float *d_fpo, float *d_po, float *d_ppo,
		      int nxpad, int nzpad) {
	
	int ix = threadIdx.x + blockIdx.x * blockDim.x;
        int iz = threadIdx.y + blockIdx.y * blockDim.y;

	if (ix < nxpad && iz < nzpad){

		int globalAddr = iz * nxpad + ix;
		
		// replace ppo with po and fpo with po
		d_ppo[globalAddr] = d_po[globalAddr];
		d_po[globalAddr] = d_fpo[globalAddr];

	}
}



__global__ void lint3d_extract_gpu(float *d_dd_pp, 
				   int it, int nr,
				   int nxpad, int nypad, int nzpad,
				   float *d_po, int *d_Rjx, int *d_Rjy, int *d_Rjz,
				   float *d_Rw000, float *d_Rw001, float *d_Rw010, float *d_Rw011, 
				   float *d_Rw100, float *d_Rw101, float *d_Rw110, float *d_Rw111) {

	int rr = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = it * nr;

	if (rr < nr){
		int y_comp = d_Rjy[rr] * nxpad * nzpad;
		int y_comp_1 = (d_Rjy[rr]+1) * nxpad * nzpad;
		int z_comp = d_Rjz[rr] * nxpad;
		int z_comp_1 = (d_Rjz[rr]+1) * nxpad;
		d_dd_pp[offset + rr] = d_po[y_comp   + z_comp   + (d_Rjx[rr])]   * d_Rw000[rr] +
                                       d_po[y_comp   + z_comp_1 + d_Rjx[rr]]     * d_Rw001[rr] +
                                       d_po[y_comp   + z_comp   + (d_Rjx[rr]+1)] * d_Rw010[rr] +
                                       d_po[y_comp   + z_comp_1 + (d_Rjx[rr]+1)] * d_Rw011[rr] +
                                       d_po[y_comp_1 + z_comp   + (d_Rjx[rr])]   * d_Rw100[rr] +
                                       d_po[y_comp_1 + z_comp_1 + d_Rjx[rr]]     * d_Rw101[rr] +
                                       d_po[y_comp_1 + z_comp   + (d_Rjx[rr]+1)] * d_Rw110[rr] +
                                       d_po[y_comp_1 + z_comp_1 + (d_Rjx[rr]+1)] * d_Rw111[rr];

	}

}


__global__ void freeSurf(float *d_po, int nrapad, int nthpad, int nb) {

        int ira = threadIdx.x + blockIdx.x * blockDim.x;
        int ith = threadIdx.y + blockIdx.y * blockDim.y;

	// apply freesurface on the extent of the planet
	// AKA where radius is greatest
	if (ith < nthpad && ira > nrapad - nb) {
		int addr = ith * nrapad + ira;
		d_po[addr] = 0;
	}
}


__global__ void onewayBC(float *uo, float *um,
	                 float *d_bzl, float *d_bzh, float *d_bxl, float *d_bxh,
		         int nxpad, int nzpad) {

	int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int iz = threadIdx.y + blockIdx.y * blockDim.y;
	int iop;

	int addr  = iz * nxpad + ix; 

	if (ix < nxpad && iz < nxpad) {

		for (ix=0; ix<nxpad; ix++) {
			for (iop=0; iop<NOP; iop++) {
		
				// top bc
				if (iz == NOP-iop) {
					uo[addr] =  um[(iz+1)*nxpad+ix] + 
						   (um[addr] - uo[(iz+1)*nxpad+ix]) * d_bzl[ix];
				}
				// bottom bc
				if (iz == nzpad-NOP+iop-1) {
					uo[addr] =  um[(iz-1)*nxpad+ix] +
						   (um[addr] - uo[(iz-1)*nxpad+ix]) * d_bzh[ix];
				}
			}
		}

		for (iz=0; iz<nzpad; iz++) {
			for (iop=0; iop<NOP; iop++) {
				
				// left bc
				if (ix == NOP-iop) {
					uo[addr] =  um[iz*nxpad+(ix+1)] + 
						   (um[addr] - uo[iz*nxpad+ix+1]) * d_bxl[iz];
				}
				// right bc
				if (ix == nxpad-NOP+iop-1) {
					uo[addr] =  um[iz*nxpad+(ix-1)] +
						   (um[addr] - uo[iz*nxpad+ix-1]) * d_bxh[iz];
				}
			}
		}

	}
}	


__global__ void spongeKernel(float *d_po, int nxpad, int nzpad, int nb){

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int z = threadIdx.y + blockIdx.y * blockDim.y;

	float alpha = 0.90;
	double damp;
	int i = 1;

	// apply sponge
	if (x < nxpad && z < nzpad) {
        
		int addr = z * nxpad + x;

		// apply to low values
		if (x < nb || z < nb){
			
			if (x < nb) { i = nb - x; }
			else { i = nb - z; }

			float fb = i / (sqrt(2.0)*(4.0*nb));
			damp = exp(-fb * fb);
			damp = exp(-1.0*fabs((pow((i-1.0),2)*log(alpha))/(pow(nb,2))));	
			d_po[addr] *= damp;
		
		}
		// apply to high values
		// NOTE: even though this is applied to all surfaces it only influences
		//       high th due to high ra being a free surface
		else if (x > nxpad - nb || z > nzpad - nb) {
				
			if (x > nxpad - nb) { i = x - (nxpad - nb); }
			else { i = z - (nzpad - nb); }
			
			float fb = i / (sqrt(2.0)*(4.0*nb));
			damp = exp(-fb * fb);
			damp = exp(-1.0*fabs((pow((i-1.0),2)*log(alpha))/(pow(nb,2))));
			d_po[addr] *= damp;

		}

	}

}


__global__ void extract(float *d_dd_pp, 
			int it, int nr,
			int nrapad, int nthpad,
			float *d_po, int *d_Rjra, int *d_Rjth,
			float *d_Rw00, float *d_Rw01, float *d_Rw10, float *d_Rw11) {

	// receiver number
	int rr = threadIdx.x + blockIdx.x * blockDim.x;
	// time offset
	// avoids rewriting over previously received data
	int offset = it * nr;

	// only perform if the receiver number represents an actual existing receiver
	if (rr < nr){

		int th_comp   = (d_Rjth[rr]) * nrapad;
		int th_comp_1 = (d_Rjth[rr]+1) * nrapad;

		// set recived pressure vals
		
		d_dd_pp[offset + rr] = d_po[th_comp   + (d_Rjra[rr])]   * d_Rw00[rr] +
                                       d_po[th_comp_1 + (d_Rjra[rr])]   * d_Rw01[rr] +
                                       d_po[th_comp   + (d_Rjra[rr]+1)] * d_Rw10[rr] +
                                       d_po[th_comp_1 + (d_Rjra[rr]+1)] * d_Rw11[rr];


	}

}
