#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

extern "C" {
    #include <rsf.h>
    #include <rsf_su.h>
}

#include "fdutil.c"

#include "cart_kernels.cu"

#define MIN(x, y) (((x) < (y)) ? (x): (y))
#define NOP 4

#ifdef _OPENMP
#include <omp.h>
#include "omputil.h"
#endif

// funct to check gpu error
static void sf_check_gpu_error (const char *msg) {
    hipError_t err = hipGetLastError ();
    if (hipSuccess != err)
	sf_error ("Cuda error: %s: %s", msg, hipGetErrorString (err));
}

// entry
int main(int argc, char*argv[]) {

    // define input variables from sconstruct
    bool fsrf, snap, bnds, dabc;
    int jsnap, jdata;

    // define IO files
    sf_file Fwav=NULL; //wavelet
    sf_file Fsou=NULL; //source
    sf_file Frec=NULL; //receivers
    sf_file Fvel=NULL; //velocity
    sf_file Fdat=NULL; //data
    sf_file Fwfl=NULL;

    // define axis
    sf_axis at, ax, az, acx, acz; // time, xdius, zeta
    sf_axis as, ar;

    // define dimension sizes
    int nt, nx, nz, ns, nr, ncs, nb;
    int it;
    float dt, dx, dz;
    float ot, ox, oz;

    // FDM structure
    fdm2d fdm=NULL;

    // device and host velocity
    float *h_vel, *d_vel;
    // pressure
    float *h_po, *d_fpo, *d_po, *d_ppo; // future, present, past
    
    float **oslice=NULL;
    float **po=NULL;

    // linear interpolation of weights and indicies
    lint2d cs, cr;

    sf_init(argc, argv);

    // exec flags
    if(! sf_getbool("free",&fsrf)) fsrf=false; /* free surface flag */
    if(! sf_getbool("dabc",&dabc)) dabc=false; /* absorbing BC */
    if(! sf_getbool("snap",&snap)) snap=true;
    if(! sf_getbool("bnds",&bnds)) bnds=true;
    sf_warning("Free Surface: %b", fsrf);
    sf_warning("Absorbing Boundaries: %b", dabc);

    // IO
    Fwav = sf_input("in");
    Fvel = sf_input("vel");
    Fsou = sf_input("sou");
    Frec = sf_input("rec");
    Fdat = sf_output("out");
    Fwfl = sf_output("wfl");

    // define gpu to be used
    int gpu;
    if (! sf_getint("gpu", &gpu)) gpu = 0;
    sf_warning("Using gpu #%d", gpu);
    hipSetDevice(gpu);

    // set up axis
    at  = sf_iaxa(Fwav,2); sf_setlabel(at ,"t" ); // time
    ax = sf_iaxa(Fvel,2); sf_setlabel(ax,"x"); // xdius
    az = sf_iaxa(Fvel,1); sf_setlabel(az,"z"); // zeta

    as  = sf_iaxa(Fsou,2); sf_setlabel(as ,"s" ); // sources
    ar  = sf_iaxa(Frec,2); sf_setlabel(ar ,"r" ); // receivers
    sf_axis ar_3, as_3;
    ar_3 = sf_iaxa(Frec, 3);
    as_3 = sf_iaxa(Fsou, 3);

    nt  = sf_n(at ); dt  = sf_d(at );
    nx = sf_n(ax); dx = sf_d(ax);
    nz = sf_n(az); dz = sf_d(az);
    
    ns  = sf_n(as_3) * sf_n(as);
    nr  = sf_n(ar_3) * sf_n(ar);

    ox = sf_o(ax); oz = sf_o(az); ot = sf_o(at);

    sf_warning("nx:%d|nz:%d|nt:%d|ns:%d|nr:%d",nx,nz,nt,ns,nr);
    sf_warning("dx:%f|dz:%f|dt:%f", dx, dz, dt);
    sf_warning("ox:%f|oz:%f|ot:%f", ox, oz, ot);
    
    // how often to extxct receiver data?
    if(! sf_getint("jdata",&jdata)) jdata=1;

    // how many time steps in each extraction?
    int nsmp = (nt/jdata);
    sf_warning("reading receiver data %d times", nsmp);

    if(! sf_getint("jdata",&jdata)) jdata=1;    // extract receiver data every jdata time steps
   
     
    // define increase in domain of model for boundary conditions
    if( !sf_getint("nb",&nb) || nb<NOP) nb=NOP;
    
    
    if(snap) {

        if(! sf_getint("jsnap",&jsnap)) jsnap=nt;       // save wavefield every jsnap time steps

	    sf_warning("Jsnap: %d", jsnap);
        acz = sf_maxa(nz,oz,dz); 
        acx = sf_maxa(nx,ox,dx); 
    
        sf_setn(at, (nt-1)/jsnap+1);
        sf_setd(at,dt*jsnap);
        
        sf_oaxa(Fwfl,acz,1);
        sf_oaxa(Fwfl,acx,2);
        sf_oaxa(Fwfl,at,3);

    }
    

    // init FDM
    fdm = fdutil_init(false, fsrf, az, ax, nb, 1);
    // origin is very slighly different under FDM due to gridsize.
    sf_warning("Adjusted Origins: oz %f, ox %f", fdm->ozpad, fdm->oxpad);
    oz = fdm->ozpad; ox = fdm->oxpad;

    // MOVE SOURCE WAVELET INTO THE GPU
    ncs = 1;
    float *ww = NULL;
    ww = sf_floatalloc(nt); // allocate var for ncs dims over nt time
    sf_floatread(ww, nt, Fwav); // read wavelet into allocated mem

    float *d_ww;
    hipMalloc((void**)&d_ww, ncs*nt*sizeof(float));
    sf_check_gpu_error("hipMalloc source wavelet to device");
    hipMemcpy(d_ww, ww, ncs*nt*sizeof(float), hipMemcpyHostToDevice);

    // SET UP SOURCE / RECEIVER COORDS
    pt2d *ss=NULL;
    pt2d *rr=NULL;

    ss = (pt2d*) sf_alloc(ns, sizeof(*ss));
    rr = (pt2d*) sf_alloc(nr, sizeof(*rr));

    float *d_Sw00, *d_Sw01, *d_Sw10, *d_Sw11;
    hipMalloc((void**)&d_Sw00, ns * sizeof(float));
    hipMalloc((void**)&d_Sw01, ns * sizeof(float));
    hipMalloc((void**)&d_Sw10, ns * sizeof(float));
    hipMalloc((void**)&d_Sw11, ns * sizeof(float));
    sf_check_gpu_error("hipMalloc source interpolation coefficients to device");

    // x and z coordinates of each source
    int *d_Sjx, *d_Sjz;
    hipMalloc((void**)&d_Sjx, ns * sizeof(int));
    hipMalloc((void**)&d_Sjz, ns * sizeof(int));
    sf_check_gpu_error("hipMalloc source coords to device");

    float *d_Rw00, *d_Rw01, *d_Rw10, *d_Rw11;
    hipMalloc((void**)&d_Rw00, nr * sizeof(float));
    hipMalloc((void**)&d_Rw01, nr * sizeof(float));
    hipMalloc((void**)&d_Rw10, nr * sizeof(float));
    hipMalloc((void**)&d_Rw11, nr * sizeof(float));
    sf_check_gpu_error("hipMalloc receiver interpolation coefficients to device");

    // x and z locations of each receiver
    int *d_Rjx, *d_Rjz;
    hipMalloc((void**)&d_Rjx, nr * sizeof(int));
    hipMalloc((void**)&d_Rjz, nr * sizeof(int));
    sf_check_gpu_error("hipMalloc receiver coords to device");

    // allocate memory to import velocity data
    float *tt1 = (float*)malloc(nx * nz * sizeof(float));

    // x, y, z pad to nxpad, nzpad, nphpad
    int nxpad=fdm->nxpad; int nzpad=fdm->nzpad; 
    h_vel = (float*)malloc(nxpad * nzpad * sizeof(float));

    // expand dimensions to allow for absorbing boundary conditions
    sf_warning("Expanding dimensions to allocate for bound. conditions");
    sf_warning("nxpad: %d | nzpad: %d", nxpad, nzpad);
    
    // read in velocity data & expand domain
    sf_floatread(tt1, nx*nz, Fvel);
    expand_cpu_2d(tt1, h_vel, fdm->nb, nz, nzpad, nx, nxpad);
    hipMalloc((void **)&d_vel, nzpad*nxpad*sizeof(float));
    sf_check_gpu_error("allocated velocity to device");
    hipMemcpy(d_vel, h_vel, nzpad*nxpad*sizeof(float), hipMemcpyHostToDevice);
    sf_check_gpu_error("copy velocity to device");

    // CREATE DATA ARRAYS FOR RECEIVERS
    float *d_dd_pp; float *h_dd_pp;
    h_dd_pp = (float*)malloc(nsmp * nr * sizeof(float));
    hipMalloc((void**)&d_dd_pp, nsmp*nr*sizeof(float));
    sf_check_gpu_error("allocate data arrays");

    // allocate pressure arrays for past, present and future on GPU's
    hipMalloc((void**)&d_ppo, nzpad*nxpad*sizeof(float));
    hipMalloc((void**)&d_po , nzpad*nxpad*sizeof(float));
    hipMalloc((void**)&d_fpo, nzpad*nxpad*sizeof(float));
    h_po = (float*)malloc(nzpad*nxpad*sizeof(float));
    sf_check_gpu_error("allocate pressure arrays");
 
    if (snap){
        oslice = sf_floatalloc2(nz,nx);
        po = sf_floatalloc2(nzpad,nxpad);
    }

    if (bnds){
        sf_setn(acz, nzpad);
        sf_setn(acx, nxpad);
        sf_oaxa(Fwfl,acz,1);
        sf_oaxa(Fwfl,acx,2);
    }

    // SET UP ONE WAY BOUND CONDITIONS
    float *one_bzl = sf_floatalloc(nxpad);
    float *one_bzh = sf_floatalloc(nxpad);
    float *one_bxl = sf_floatalloc(nzpad);
    float *one_bxh = sf_floatalloc(nzpad);

    float d;
    for (int ix=0; ix<nxpad; ix++) {
	d = h_vel[NOP * nxpad + ix] * (dt / dz);
	one_bzl[ix] = (1-d)/(1+d);
	d = h_vel[(nzpad-NOP-1)*nxpad + ix] * (dt / dz);
	one_bzh[ix] = (1-d)/(1+d);
    }
    for (int iz=0; iz<nzpad; iz++) {
        d = h_vel[iz * nxpad + NOP] * (dt / dx);
	one_bxl[iz] = (1-d)/(1+d);
	d = h_vel[iz * nxpad + nxpad-NOP-1] * (dt / dx);
	one_bxh[iz] = (1-d)/(1+d);
    }

    float *d_bzl, *d_bzh, *d_bxl, *d_bxh;
    hipMalloc((void**)&d_bzl, nxpad*sizeof(float));
    hipMemcpy(d_bzl, one_bzl, nxpad*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_bzh, nxpad*sizeof(float));
    hipMemcpy(d_bzh, one_bzh, nxpad*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_bxl, nzpad*sizeof(float));
    hipMemcpy(d_bxl, one_bxl, nzpad*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_bxh, nzpad*sizeof(float));
    hipMemcpy(d_bxh, one_bxh, nzpad*sizeof(float), hipMemcpyHostToDevice);

    // ITERATE OVER SHOTS
    for (int isrc = 0; isrc < 1; isrc ++) {

	// read source and receiver coordinates
	pt2dread1(Fsou, ss, ns, 2);
	pt2dread1(Frec, rr, nr, 2);

	// set source on GPU
	sf_warning("Source location: ");
	printpt2d(*ss);

	// perform 3d linear interpolation on source
	cs = lint2d_make(ns, ss, fdm);

	sf_warning("Source interp coeffs:");
    sf_warning("00:%f | 01:%f | 10:%f | 11:%f", cs->w00[0], cs->w01[0], cs->w10[0], cs->w11[0]);

    hipMemcpy(d_Sw00, cs->w00, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw01, cs->w01, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw10, cs->w10, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Sw11, cs->w11, ns * sizeof(float), hipMemcpyHostToDevice);
    sf_check_gpu_error("copy source interpolation coefficients to device");

    hipMemcpy(d_Sjz, cs->jz, ns * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_Sjx, cs->jx, ns * sizeof(int), hipMemcpyHostToDevice);
	sf_check_gpu_error("copy source coords to device");


	// SET RECEIVERS ON THE GPU
	sf_warning("Receiver Count: %d", nr);
	cr = lint2d_make(nr, rr, fdm);

	hipMemcpy(d_Rw00, cr->w00, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw01, cr->w01, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw10, cr->w10, nr * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Rw11, cr->w11, nr * sizeof(float), hipMemcpyHostToDevice);
    sf_check_gpu_error("copy receiver interpolation coefficients to device");

    hipMemcpy(d_Rjz, cr->jz, nr * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Rjx, cr->jx, nr * sizeof(int), hipMemcpyHostToDevice);
    sf_check_gpu_error("copy receiver coords to device");


	// set pressure to 0 on gpu
	hipMemset(d_ppo, 0, nzpad*nxpad*sizeof(float));
	hipMemset(d_po , 0, nzpad*nxpad*sizeof(float));
	hipMemset(d_fpo, 0, nzpad*nxpad*sizeof(float));
	sf_check_gpu_error("Set pressure arrays to 0");

	// set receiver data to 0
	hipMemset(d_dd_pp, 0, nsmp*nr*sizeof(float));

	for (int i=0; i < nsmp*nr; i++) {
	    h_dd_pp[i] = 0.f;
	}

	// TIME LOOP
	fprintf(stderr,"total num of time steps: %d \n", nt);

	for (it=0; it<nt; it++) {

	    fprintf(stderr, "\b\b\b\b\b\b\b\b\b\b\b\b\b\b\btime step: %d", it+1);

	    // INJECT PRESSURE SOURCE
        dim3 dimGridS(MIN(ns, ceil(ns/1024.0f)), 1);
        dim3 dimBlockS(MIN(ns, 1024), 1);
        inject_sources_2D<<<dimGridS,dimBlockS>>>(d_po, d_ww, d_vel,
                       d_Sw00, d_Sw01, d_Sw10, d_Sw11,
                       d_Sjx, d_Sjz, it, ns, fdm->nxpad, fdm->nzpad);
        sf_check_gpu_error("inject sources Kernel");

	    // APPLY WAVE EQUATION
	    dim3 dimGrid2(ceil(nxpad/8.0f),ceil(nzpad/8.0f));
	    dim3 dimBlock2(8,8);
	    solve_2D<<<dimGrid2, dimBlock2>>>(d_fpo, d_po, d_ppo,
			    		  d_vel,
					  dx, dz, dt,
					  nxpad, nzpad);
	    sf_check_gpu_error("solve Kernel");

	    // SHIFT PRESSURE FIELDS IN TIME
	    shift_2D<<<dimGrid2, dimBlock2>>>(d_fpo, d_po, d_ppo,
					   nxpad, nzpad);
	    sf_check_gpu_error("shift Kernel");

	    // ONE WAY BC
        onewayBC_2D<<<dimGrid2, dimBlock2>>>(d_po, d_ppo,
                        d_bzl, d_bzh, d_bxl, d_bxh,
                        nxpad, nzpad);

	    // SPONGE
	    spongeKernel_2D<<<dimGrid2, dimBlock2>>>(d_po, nxpad, nzpad, nb);
	    sf_check_gpu_error("sponge Kernel");
	    spongeKernel_2D<<<dimGrid2, dimBlock2>>>(d_ppo, nxpad, nzpad, nb);
        sf_check_gpu_error("sponge Kernel");

	    // FREE SURFACE
	    freeSurf_2D<<<dimGrid2, dimBlock2>>>(d_po, nxpad, nzpad, nb);
	    sf_check_gpu_error("freeSurf Kernel");

	    if (snap && it%jsnap==0) {

            hipMemcpy(h_po, d_po, nxpad*nzpad*sizeof(float), hipMemcpyDefault);

#ifdef _OPENMP
#pragma omp parallel for schedule(dynamic) private(x, z) shared(po, h_po, nxpad)
#endif

            for (int x = 0; x < nxpad; x++) {
                for (int z = 0; z < nzpad; z++) {
                    po[x][z] = h_po[z*nxpad + x];
                }
            }

            if (bnds) {
                sf_floatwrite(po[0], nzpad*nxpad, Fwfl);
            }
            else {

                cut2d(po, oslice, fdm, az, ax);

                sf_floatwrite(oslice[0], sf_n(az)*sf_n(ax), Fwfl);

            }
	    }
	    
	    // EXTRACT TO RECEIVERS
	    dim3 dimGrid3(MIN(nr, ceil(nr/1024.0f)), 1);
	    dim3 dimBlock3(MIN(nr, 1024), 1);

	    extract_2D<<<dimGrid3, dimBlock3>>>(d_dd_pp, it, nr,
			    		     nxpad, nzpad, 
					     d_po, d_Rjx, d_Rjz,
					     d_Rw00, d_Rw01, d_Rw10, d_Rw11);
	    sf_check_gpu_error("extract Kernel");

	}

    }

    fprintf(stderr,"\n");

  
    hipMemcpy(h_dd_pp, d_dd_pp, nsmp*nr*sizeof(float), hipMemcpyDefault);

    sf_setn(ar, nr);
    sf_setn(at, nsmp);
    sf_setd(at, dt*jdata);

    sf_oaxa(Fdat, at, 2);
    sf_oaxa(Fdat, ar, 1);

    sf_floatwrite(h_dd_pp, nsmp*nr, Fdat);    
    
    // FREE ALLOCATED MEMORY
    hipFree(d_ww);

    hipFree(d_Sw00); hipFree(d_Sw01); hipFree(d_Sw10); hipFree(d_Sw11);
    hipFree(d_Sjx); hipFree(d_Sjz);
    
    hipFree(d_Rw00); hipFree(d_Rw01); hipFree(d_Rw10); hipFree(d_Rw11);
    hipFree(d_Rjx); hipFree(d_Rjz);

    hipFree(d_dd_pp);
    hipFree(d_ppo); hipFree(d_po); hipFree(d_fpo);

}

