
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

// macro for 1d index to simulate a 3d matrix
#define INDEX2D(ix, iz, nx) ((ix)+(iz)*(nx))
// PLACEHOLDER
#define INDEX3D(ix, iy, iz, nx, nz) ((ix)+(iz)*(nx)+(iy)*(nx)*(nz))

void expand_cpu_2d(float *a, float *b, int nb, int x_a, int x_b, int z_a, int z_b){

        // copy into other array
	for (int ix = 0; ix < x_a; ix++) {
                for (int iz = 0; iz < z_a; iz++) {
                        b[INDEX2D(ix+nb,iz+nb,x_b)] = a[INDEX2D(ix,iz,x_a)];
		}
        }
	
        // expand z direction
	for (int ix = 0; ix < x_b; ix++) {
                for (int iz = 0; iz < nb; iz++) {
                        b[INDEX2D(ix,iz,x_b)] = b[INDEX2D(ix,nb,x_b)];
                        b[INDEX2D(ix,z_b-iz-1,x_b)] = b[INDEX2D(ix,z_b-nb-1,x_b)];
                }
        }
	
        //expand x direction 
	for (int ix = 0; ix < nb; ix++) {
                for (int iz = 0; iz < z_b; iz++) {
                        b[INDEX2D(ix,iz,x_b)] = b[INDEX2D(nb,iz,x_b)];
                        b[INDEX2D(x_b-ix-1,iz,x_b)] = b[INDEX2D(x_b-nb-1,iz,x_b)];
                }
        }
	
}

__global__ void lint2d_bell_gpu(float *d_uu, float *d_ww, float *d_Sw00, float *d_Sw01, float *d_Sw10, float *d_Sw11, float *d_bell, int *d_jx, int *d_jz, int it, int nc, int ns, int c, int nbell, int nxpad) {

        int ix = threadIdx.x;
        int iz = threadIdx.y;
        int ia = blockIdx.x;

        float wa = d_ww[it * nc * ns + c * ns + ia] * d_bell[(iz * (2*nbell+1)) + ix];

	int z_comp = (d_jz[ia] - nbell) + iz;
	int x_comp = (d_jx[ia] - nbell) + ix;

        atomicAdd(&d_uu[(z_comp)     * nxpad + (x_comp    )], ((wa * d_Sw00[ia])));
        atomicAdd(&d_uu[(z_comp + 1) * nxpad + (x_comp    )], ((wa * d_Sw01[ia])));
        atomicAdd(&d_uu[(z_comp)     * nxpad + (x_comp + 1)], ((wa * d_Sw10[ia])));
        atomicAdd(&d_uu[(z_comp + 1) * nxpad + (x_comp + 1)], ((wa * d_Sw11[ia])));

}


// divergence 3d for cpml
#define NOP 4 // half of the order in space

__global__ void solve(float *d_fpo, float *d_po, float *d_ppo, float *d_vel,
		      float dra, float dth, float ora, float oth, float dt,
		      int nrapad, int nthpad) {

	int ira = threadIdx.x + blockIdx.x * blockDim.x;
	int ith = threadIdx.y + blockIdx.y * blockDim.y;

	if (ira < nrapad && ith < nthpad){
		
		int globalAddr = ith * nrapad + ira;			  
		float laplace;
		float compra, compth;

		// extract true location from deltas and indicies
		float ra; float th;
		ra = dra * ira + ora;
		th = dth * ith + oth;
		
		// extract true velocity
		float v;
		v  = d_vel[globalAddr];

		// perform only in boundaries:
		if (ira >= NOP && ira < nrapad-NOP && ith >= NOP && ith < nthpad - NOP) {

			// CALC LAPLACE VIA STENCIL
			
			// START BY LOOKING AT VALS ALONG -R then R then +R
			compra = ((1/(dra*dra))+(1/(2*ra*dra))) * d_po[INDEX2D(ira-1,ith,nrapad)] + 
				 (-2/(dra*dra))                 * d_po[globalAddr] +
				 ((1/(dra*dra))-(1/(2*ra*dra))) * d_po[INDEX2D(ira+1,ith,nrapad)];
			
			// NOW COMPUTE COMPONENTS DEPENDENT ON THETA
			compth = ((1/(ra*ra*dth*dth))) * d_po[INDEX2D(ira,ith-1,nrapad)] + 
				 (-2/(ra*ra*dth*dth))  * d_po[globalAddr] + 
				 ((1/(ra*ra*dth*dth))) * d_po[INDEX2D(ira,ith+1,nrapad)];

			// SUM TO GET LAPLACIAN
			laplace = compra + compth;

		} else {
			laplace = 0.;
		}

		// compute pressure at next time step
		d_fpo[globalAddr] = (dt*dt) * (v*v) * laplace + 2*d_po[globalAddr] - d_ppo[globalAddr];


	}

}


__global__ void shift(float *d_fpo, float *d_po, float *d_ppo,
		      int nrapad, int nthpad) {
	
	int ira = threadIdx.x + blockIdx.x * blockDim.x;
        int ith = threadIdx.y + blockIdx.y * blockDim.y;

	if (ira < nrapad && ith < nthpad){

		int globalAddr = ith * nrapad + ira;
		
		// replace ppo with po and fpo with po
		d_ppo[globalAddr] = d_po[globalAddr];
		d_po[globalAddr] = d_fpo[globalAddr];

	}
}



__global__ void lint3d_extract_gpu(float *d_dd_pp, 
				   int it, int nr,
				   int nxpad, int nypad, int nzpad,
				   float *d_po, int *d_Rjx, int *d_Rjy, int *d_Rjz,
				   float *d_Rw000, float *d_Rw001, float *d_Rw010, float *d_Rw011, 
				   float *d_Rw100, float *d_Rw101, float *d_Rw110, float *d_Rw111) {

	int rr = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = it * nr;

	if (rr < nr){
		int y_comp = d_Rjy[rr] * nxpad * nzpad;
		int y_comp_1 = (d_Rjy[rr]+1) * nxpad * nzpad;
		int z_comp = d_Rjz[rr] * nxpad;
		int z_comp_1 = (d_Rjz[rr]+1) * nxpad;
		d_dd_pp[offset + rr] = d_po[y_comp   + z_comp   + (d_Rjx[rr])]   * d_Rw000[rr] +
                                       d_po[y_comp   + z_comp_1 + d_Rjx[rr]]     * d_Rw001[rr] +
                                       d_po[y_comp   + z_comp   + (d_Rjx[rr]+1)] * d_Rw010[rr] +
                                       d_po[y_comp   + z_comp_1 + (d_Rjx[rr]+1)] * d_Rw011[rr] +
                                       d_po[y_comp_1 + z_comp   + (d_Rjx[rr])]   * d_Rw100[rr] +
                                       d_po[y_comp_1 + z_comp_1 + d_Rjx[rr]]     * d_Rw101[rr] +
                                       d_po[y_comp_1 + z_comp   + (d_Rjx[rr]+1)] * d_Rw110[rr] +
                                       d_po[y_comp_1 + z_comp_1 + (d_Rjx[rr]+1)] * d_Rw111[rr];

	}

}


__global__ void freeSurf(float *d_po, int nrapad, int nthpad, int nb) {

        int ira = threadIdx.x + blockIdx.x * blockDim.x;
        int ith = threadIdx.y + blockIdx.y * blockDim.y;

	// apply freesurface on the extent of the planet
	// AKA where radius is greatest
	if (ith < nthpad && ira > nrapad - nb) {
		int addr = ith * nrapad + ira;
		d_po[addr] = 0;
	}
}


__global__ void spongeKernel(float *d_po, int nxpad, int nzpad, int nb){

        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int z = threadIdx.y + blockIdx.y * blockDim.y;

        float alpha = 0.90;
        double damp;
        int i = 1;

        // apply sponge
        if (x < nxpad && z < nzpad) {

                int addr = z * nxpad + x;

                // apply to low values
                if (x < nb || z < nb){

                        if (x < nb) { i = nb - x; }
                        else { i = nb - z; }

                        float fb = i / (sqrt(2.0)*(4.0*nb));
                        damp = exp(-fb * fb);
                        damp = exp(-1.0*fabs((pow((i-1.0),2)*log(alpha))/(pow(nb,2))));
                        d_po[addr] *= damp;

                }
                // apply to high values
                // NOTE: even though this is applied to all surfaces it only influences
                //       high th due to high ra being a free surface
                else if (x > nxpad - nb || z > nzpad - nb) {

                        if (x > nxpad - nb) { i = x - (nxpad - nb); }
                        else { i = z - (nzpad - nb); }

                        float fb = i / (sqrt(2.0)*(4.0*nb));
                        damp = exp(-fb * fb);
                        damp = exp(-1.0*fabs((pow((i-1.0),2)*log(alpha))/(pow(nb,2))));
                        d_po[addr] *= damp;

                }

        }

}


__global__ void spongeKernelOLD(float *d_po, int nrapad, int nthpad, int nb){

	int ra = threadIdx.x + blockIdx.x * blockDim.x;
	int th = threadIdx.y + blockIdx.y * blockDim.y;

	float alpha = 0.95;
	double damp;
	int i = 1;

	// apply sponge
	if (ra < nrapad && th < nthpad) {
        
		int addr = th * nrapad + ra;

		// apply to low values
		if (ra < nb || th < nb){
			
			if (ra < nb) { i = nb - ra; }
			else { i = nb - th; }

			// dampining funct 1
			//double damp = exp(-1.0*fabs(((i-1.0)*log(alpha))/nb)); 
			
			// dampining funct 2
			//double damp = exp(-1.0*fabs((pow((i-1.0),2)*log(alpha))/(pow(nb,2))));

			//double damp = 0.5 + 0.5*cos(((1.0*i)/nb)*(3.14159));

			float fb = i / (sqrt(2.0)*(4.0*nb));
			damp = exp(-fb * fb);	
			//d_po[addr] = damp;
			d_po[addr] *= damp;
		
		}
		// apply to high values
		// NOTE: even though this is applied to all surfaces it only influences
		//       high th due to high ra being a free surface
		else if (ra > nrapad - nb || th > nthpad - nb) {
				
			if (ra > nrapad - nb) { i = ra - (nrapad - nb); }
			else { i = th - (nthpad - nb); }
			
			// dampining funct 1
			//double damp = exp(-1.0*fabs(((i-1.0)*log(alpha))/nb));

                        // dampining funct 2
                        double damp = exp(-1.0*fabs((pow((i-1.0),2)*log(alpha))/(pow(nb,2))));
			
			//double damp = 0.5 + 0.5*cos(((1.0*i)/nb)*(3.14159));
			
			float fb = i / (sqrt(2.0)*(4.0*nb));
			damp = exp(-fb * fb);
			//d_po[addr] = damp;
			d_po[addr] *= damp;

		}

		//else { d_po[addr] = 1; }

	}

}


__global__ void onewayBC(float *uo, float *um,
		         float *d_bthl, float *d_bthh, float *d_bral, float *d_brah,
			 int nrapad, int nthpad) {

	int ira = threadIdx.x + blockIdx.x * blockDim.x;
	int ith = threadIdx.y + blockIdx.y * blockDim.y;
	int iop;

	int addr = ith * nrapad + ira;

	if (ira < nrapad && ith < nthpad) {

		for (ira=0; ira<nrapad; ira++) {
			for (iop=0; iop<NOP; iop++) {
				
				// top bc
				if (ith == NOP-iop) {
					uo[addr] =  um[(ith+1)*nrapad+ira] +
						   (um[addr] - uo[(ith+1)*nrapad+ira]) * d_bthl[ira];
				}
				// bottom bc
				if (ith == nthpad-NOP+iop-1) {
                                        uo[addr] =  um[(ith-1)*nrapad+ira] +
                                                   (um[addr] - uo[(ith-1)*nrapad+ira]) * d_bthh[ira];
                                }

			}
		}

		for (ith=0; ith<nthpad; ith++) {
			for (iop=0; iop<NOP; iop++) {
				
				// left bc
				if (ira == NOP-iop) {
					uo[addr] =  um[ith*nrapad+ira+1] +
                                                   (um[addr] - uo[ith*nrapad+ira+1]) * d_bral[ith];
				}
				// bottom bc
				if (ira == nrapad-NOP+iop-1) {
					uo[addr] =  um[ith*nrapad+ira-1] +
                                                   (um[addr] - uo[ith*nrapad+ira-1]) * d_brah[ith];
				}

			}
		}		

	}

}


__global__ void extract(float *d_dd_pp, 
			int it, int nr,
			int nrapad, int nthpad,
			float *d_po, int *d_Rjra, int *d_Rjth,
			float *d_Rw00, float *d_Rw01, float *d_Rw10, float *d_Rw11) {

	// receiver number
	int rr = threadIdx.x + blockIdx.x * blockDim.x;
	// time offset
	// avoids rewriting over previously received data
	int offset = it * nr;

	// only perform if the receiver number represents an actual existing receiver
	if (rr < nr){

		int th_comp   = (d_Rjth[rr]) * nrapad;
		int th_comp_1 = (d_Rjth[rr]+1) * nrapad;

		// set recived pressure vals
		
		d_dd_pp[offset + rr] = d_po[th_comp   + (d_Rjra[rr])]   * d_Rw00[rr] +
                                       d_po[th_comp_1 + (d_Rjra[rr])]   * d_Rw01[rr] +
                                       d_po[th_comp   + (d_Rjra[rr]+1)] * d_Rw10[rr] +
                                       d_po[th_comp_1 + (d_Rjra[rr]+1)] * d_Rw11[rr];


	}

}
